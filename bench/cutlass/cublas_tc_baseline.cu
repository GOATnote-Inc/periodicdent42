// cuBLAS Tensor Core baseline for Q@K^T
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <stdio.h>

int main() {
    const int M = 32, N = 32, K = 64;
    
    printf("cuBLAS TensorCore Baseline\n");
    printf("M=%d, N=%d, K=%d\n", M, N, K);
    
    // Allocate
    half *h_A = (half*)malloc(M * K * sizeof(half));
    half *h_B = (half*)malloc(K * N * sizeof(half));
    float *h_C = (float*)malloc(M * N * sizeof(float));
    
    for (int i = 0; i < M * K; i++) h_A[i] = __float2half(0.1f);
    for (int i = 0; i < K * N; i++) h_B[i] = __float2half(0.1f);
    for (int i = 0; i < M * N; i++) h_C[i] = 0.0f;
    
    half *d_A, *d_B;
    float *d_C;
    hipMalloc(&d_A, M * K * sizeof(half));
    hipMalloc(&d_B, K * N * sizeof(half));
    hipMalloc(&d_C, M * N * sizeof(float));
    
    hipMemcpy(d_A, h_A, M * K * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(half), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(float), hipMemcpyHostToDevice);
    
    // cuBLAS setup
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Enable TensorCore (default on Ada)
    hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
    
    // C = alpha * A @ B^T + beta * C
    // A: M×K row-major → K×M col-major (transposed)
    // B: K×N row-major → N×K col-major (transposed)
    // Result: M×N row-major → N×M col-major
    
    float alpha = 1.0f / sqrtf(K);
    float beta = 0.0f;
    
    // GemmEx: FP16 input, FP32 output, FP32 compute
    hipblasStatus_t status = hipblasGemmEx(
        handle,
        HIPBLAS_OP_T,  // B transposed
        HIPBLAS_OP_T,  // A transposed
        N, M, K,      // Swapped for col-major
        &alpha,
        d_B, HIP_R_16F, K,  // B
        d_A, HIP_R_16F, K,  // A
        &beta,
        d_C, HIP_R_32F, N,  // C
        HIPBLAS_COMPUTE_32F,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP
    );
    
    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "❌ hipblasGemmEx failed: %d\n", status);
        return 1;
    }
    
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "❌ CUDA error: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    printf("✅ cuBLAS launch success\n");
    
    // Verify
    hipMemcpy(h_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    
    float expected = 0.1f * 0.1f * K * alpha;
    printf("C[0,0]=%.6f (expect ~%.6f)\n", h_C[0], expected);
    
    bool ok = (h_C[0] > 0.0f && h_C[0] < 2.0f * expected);
    printf("%s\n", ok ? "✅ CORRECT" : "❌ WRONG");
    
    // Benchmark
    if (ok) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        const int warmup = 10, iters = 100;
        for (int i = 0; i < warmup; i++) {
            hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, M, K,
                         &alpha, d_B, HIP_R_16F, K, d_A, HIP_R_16F, K,
                         &beta, d_C, HIP_R_32F, N,
                         HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        hipDeviceSynchronize();
        
        hipEventRecord(start);
        for (int i = 0; i < iters; i++) {
            hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, M, K,
                         &alpha, d_B, HIP_R_16F, K, d_A, HIP_R_16F, K,
                         &beta, d_C, HIP_R_32F, N,
                         HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        printf("\n✅ cuBLAS TC Baseline: %.2f μs/iter\n", ms * 1000.0f / iters);
        printf("   (This is reference speed for Q@K^T with Tensor Cores)\n");
        
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    
    // Cleanup
    hipblasDestroy(handle);
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    
    return ok ? 0 : 1;
}

