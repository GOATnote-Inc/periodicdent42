#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <cstdint>
#include <cmath>
#include <cstdio>

using namespace nvcuda;

// Debug flag: enable with -DDEBUG_PRINT during compilation
// #define DEBUG_PRINT 1

// --- Tunables (L4 sm_89, full WMMA) ---
#define HEAD_DIM 64
#define TILE_M   32      // Q rows per block (2 WMMA tiles)
#define TILE_N   32      // KV rows per tile (2 WMMA tiles)
#define NUM_WARPS 4      // REDUCED for WMMA (each warp handles one 16×16 tile)
#define THREADS_PER_BLOCK (NUM_WARPS * 32)
#define D_PAD    64      // No padding needed for WMMA (64 is 16-aligned)

// WMMA tile size
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// Warp reductions (FP32)
__device__ __forceinline__ float warp_reduce_sum(float v){
    #pragma unroll
    for (int o=16; o>0; o>>=1) v += __shfl_down_sync(0xffffffff, v, o);
    return v;
}

__device__ __forceinline__ float warp_reduce_max(float v){
    #pragma unroll
    for (int o=16; o>0; o>>=1) v = fmaxf(v, __shfl_down_sync(0xffffffff, v, o));
    return v;
}

// Sim-FP8 dequant (symmetric, zero maps exactly to 0)
__device__ __forceinline__ float dequant_sim_fp8(uint8_t u, float s){
    constexpr float INV_MAX = 1.0f / 127.0f;
    float centered = (static_cast<float>(static_cast<int>(u) - 128)) * INV_MAX;
    return centered * 448.0f * s;
}

__launch_bounds__(THREADS_PER_BLOCK, 4)
__global__ void sdpa_fp8_stage_c_wmma_kernel(
    const uint8_t* __restrict__ Q,   // [B,H,S,D]
    const uint8_t* __restrict__ K,   // [B,H,S,D]
    const uint8_t* __restrict__ V,   // [B,H,S,D]
    const float*   __restrict__ Qs,  // [H]
    const float*   __restrict__ Ks,  // [H]
    const float*   __restrict__ Vs,  // [H]
    half*          __restrict__ O,   // [B,H,S,D]
    int B, int H, int S, int D, float softmax_scale
){
    const int b = blockIdx.z;
    const int h = blockIdx.y;
    const int q_block = blockIdx.x;
    const int tid = threadIdx.x;
    const int warp_id = tid >> 5;
    const int lane    = tid & 31;

    const int q_start = q_block * TILE_M;
    const int q_end   = min(q_start + TILE_M, S);
    const int rows_in_tile = q_end - q_start;
    if (rows_in_tile <= 0) return;

    const size_t BHSD = (size_t)S * D;
    const uint8_t* Qbh = Q + ((size_t)b * H + h) * BHSD;
    const uint8_t* Kbh = K + ((size_t)b * H + h) * BHSD;
    const uint8_t* Vbh = V + ((size_t)b * H + h) * BHSD;
    half*         Obh  = O + ((size_t)b * H + h) * BHSD;

    const float q_s = Qs[h], k_s = Ks[h], v_s = Vs[h];

    // --- Shared memory ---
    // Q: row-major [TILE_M][D]
    // K: col-major [D][TILE_N] (transposed for WMMA)
    // V: row-major [TILE_N][D]
    __shared__ alignas(16) half sQ[TILE_M][D_PAD];     // 4 KB
    __shared__ alignas(16) half sKT[D_PAD][TILE_N];    // 4 KB (transposed!)
    __shared__ alignas(16) half sV[TILE_N][D_PAD];     // 4 KB
    
    __shared__ float kLUT[256];  // K dequant lookup
    __shared__ float vLUT[256];  // V dequant lookup
    __shared__ float m_smem[TILE_M];
    __shared__ float l_smem[TILE_M];
    __shared__ alignas(16) float U_smem[TILE_M][D_PAD];  // 8 KB

    // Total SMEM: 4+4+4+8+3 = 23 KB < 48 KB ✅

    // --- Build LUTs ---
    if (tid < 256) {
        const int u = tid;
        constexpr float INV_MAX = 1.0f / 127.0f;
        float centered = (static_cast<float>(u) - 128.0f) * INV_MAX;
        float decoded = centered * 448.0f;
        kLUT[u] = decoded * k_s;
        vLUT[u] = decoded * v_s;
    }

    // --- Load Q tile (uint8→FP16, row-major) ---
    for (int idx = tid; idx < rows_in_tile * D; idx += blockDim.x) {
        int r = idx / D;
        int d = idx % D;
        uint8_t q_u8 = Qbh[(size_t)(q_start + r) * D + d];
        float f = dequant_sim_fp8(q_u8, q_s);
        sQ[r][d] = __float2half(f);
    }

#ifdef DEBUG_PRINT
    __syncthreads();
    if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && tid == 0) {
        printf("[DEBUG] Q tile loaded (row 0, d=0:5): ");
        for (int d = 0; d < 5; d++) {
            printf("%.4f ", __half2float(sQ[0][d]));
        }
        printf("\n");
    }
#endif

    // Init stats and U
    for (int idx = tid; idx < rows_in_tile * D; idx += blockDim.x) {
        int r = idx / D;
        int d = idx % D;
        U_smem[r][d] = 0.f;
    }
    for (int r = tid; r < rows_in_tile; r += blockDim.x) {
        m_smem[r] = -INFINITY;
        l_smem[r] = 0.f;
    }
    __syncthreads();

    const int nTiles = (S + TILE_N - 1) / TILE_N;

    for (int t = 0; t < nTiles; ++t) {
        const int kv_start = t * TILE_N;
        const int kv_end   = min(kv_start + TILE_N, S);
        const int kv_len   = kv_end - kv_start;

        // --- Load K tile (uint8→FP16, TRANSPOSED to col-major) ---
        // sKT[d][n] = K[n][d] (transposed for WMMA B matrix)
        for (int idx = tid; idx < kv_len * D; idx += blockDim.x) {
            int n = idx / D;
            int d = idx % D;
            uint8_t k_u8 = Kbh[(size_t)(kv_start + n) * D + d];
            sKT[d][n] = __float2half(kLUT[k_u8]);  // Transposed!
        }

        // --- Load V tile (uint8→FP16, row-major) ---
        for (int idx = tid; idx < kv_len * D; idx += blockDim.x) {
            int n = idx / D;
            int d = idx % D;
            uint8_t v_u8 = Vbh[(size_t)(kv_start + n) * D + d];
            sV[n][d] = __float2half(vLUT[v_u8]);
        }
        __syncthreads();

#ifdef DEBUG_PRINT
        if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && tid == 0 && t == 0) {
            printf("[DEBUG] V tile loaded (row 0, d=0:5): ");
            for (int d = 0; d < 5; d++) {
                printf("%.4f ", __half2float(sV[0][d]));
            }
            printf("\n");
        }
#endif

        // =========================================
        // WMMA COMPUTE: Q @ K^T → S (32×32)
        // =========================================
        // Each warp handles one 16×16 output tile
        // 4 warps cover 2×2 = 32×32 output
        
        // Warp mapping:
        // warp_id=0 → S[0:16,  0:16]
        // warp_id=1 → S[0:16, 16:32]
        // warp_id=2 → S[16:32, 0:16]
        // warp_id=3 → S[16:32,16:32]
        
        const int warp_m = (warp_id / 2) * WMMA_M;  // 0 or 16
        const int warp_n = (warp_id % 2) * WMMA_N;  // 0 or 16

        // WMMA fragments
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;

        // Zero accumulator
        wmma::fill_fragment(c_frag, __float2half(0.0f));

        // Compute Q@K^T in 16×16×16 chunks (4 chunks for D=64)
        #pragma unroll
        for (int k = 0; k < D; k += WMMA_K) {
            // Load A: Q[warp_m:warp_m+16, k:k+16] (row-major)
            wmma::load_matrix_sync(a_frag, &sQ[warp_m][k], D_PAD);
            
            // Load B: K^T[k:k+16, warp_n:warp_n+16] (col-major)
            wmma::load_matrix_sync(b_frag, &sKT[k][warp_n], TILE_N);
            
            // MMA: C += A * B
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }

        // Store S to shared memory temporarily for softmax
        __shared__ alignas(16) half sS[TILE_M][TILE_N];
        wmma::store_matrix_sync(&sS[warp_m][warp_n], c_frag, TILE_N, wmma::mem_row_major);
        __syncthreads();

#ifdef DEBUG_PRINT
        if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && tid == 0 && t == 0) {
            printf("[DEBUG] Q@K^T scores (row 0, n=0:5): ");
            for (int n = 0; n < 5; n++) {
                printf("%.4f ", __half2float(sS[0][n]));
            }
            printf("\n");
        }
#endif

        // =========================================
        // ONLINE SOFTMAX (per row, scalar path)
        // =========================================
        // Each warp handles 32/4 = 8 rows
        for (int r = warp_id; r < rows_in_tile; r += NUM_WARPS) {
            // PRIORITY 1 FIX: Each lane loads ALL scores (no stride, no broadcast)
            // Previous bug: Only lane N loaded S_row[N], leaving most elements uninitialized
            // Correct: Each lane loads full S_row[] sequentially → all lanes see same data
            float S_row[TILE_N];
            #pragma unroll
            for (int n = 0; n < kv_len; ++n) {
                S_row[n] = __half2float(sS[r][n]) * softmax_scale;
            }

            // Online softmax update
            float m_old = m_smem[r];
            float m_new = m_old;
            #pragma unroll
            for (int n = 0; n < kv_len; ++n) {
                m_new = fmaxf(m_new, S_row[n]);
            }

            float l_old = l_smem[r];
            float l_add = 0.f;
            #pragma unroll
            for (int n = 0; n < kv_len; ++n) {
                S_row[n] = __expf(S_row[n] - m_new);
                l_add += S_row[n];
            }

            float rescale = __expf(m_old - m_new);
            float l_new = l_old * rescale + l_add;

#ifdef DEBUG_PRINT
            if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && r == 0 && warp_id == 0 && lane == 0 && t == 0) {
                printf("[DEBUG] Softmax (row 0): m_old=%.4f m_new=%.4f l_old=%.4f l_add=%.4f rescale=%.4f\n",
                       m_old, m_new, l_old, l_add, rescale);
                printf("[DEBUG] Attention weights P[0:5]: ");
                for (int n = 0; n < 5; n++) {
                    printf("%.4f ", S_row[n]);
                }
                printf("\n");
            }
#endif

            // Scale U
            for (int d = lane; d < D; d += 32) {
                U_smem[r][d] *= rescale;
            }

            // Accumulate P·V (unnormalized)
            for (int n = 0; n < kv_len; ++n) {
                float p = S_row[n];
                for (int d = lane; d < D; d += 32) {
                    float v = __half2float(sV[n][d]);
                    U_smem[r][d] += p * v;
                }
            }

            if (lane == 0) {
                m_smem[r] = m_new;
                l_smem[r] = l_new;
            }
        }
        __syncthreads();
    }

    // --- Write O = U / l ---
    for (int r = warp_id; r < rows_in_tile; r += NUM_WARPS) {
        float l_final = l_smem[r];
        half* out = Obh + (size_t)(q_start + r) * D;
        
#ifdef DEBUG_PRINT
        if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && r == 0 && warp_id == 0 && lane == 0) {
            printf("[DEBUG] Final output (row 0): l_final=%.4f\n", l_final);
            printf("[DEBUG] U_smem[0][0:5] / l_final: ");
            for (int d = 0; d < 5; d++) {
                printf("%.4f ", U_smem[0][d] / l_final);
            }
            printf("\n");
        }
#endif
        
        for (int d = lane; d < D; d += 32) {
            float o = U_smem[r][d] / l_final;
            out[d] = __float2half(o);
        }
    }
}

extern "C" void launch_sdpa_fp8_stage_c_wmma(
    const void* Q,
    const void* K,
    const void* V,
    const float* Q_scale,
    const float* K_scale,
    const float* V_scale,
    half* O,
    int B, int H, int S, int D,
    float softmax_scale,
    hipStream_t stream
) {
    dim3 grid((S + TILE_M - 1) / TILE_M, H, B);
    dim3 block(THREADS_PER_BLOCK);

    sdpa_fp8_stage_c_wmma_kernel<<<grid, block, 0, stream>>>(
        reinterpret_cast<const uint8_t*>(Q),
        reinterpret_cast<const uint8_t*>(K),
        reinterpret_cast<const uint8_t*>(V),
        Q_scale,
        K_scale,
        V_scale,
        O,
        B, H, S, D,
        softmax_scale
    );
}

