// flashcore/fast/attention_cublas_tc.cu
// cuBLAS (non-Lt) approach - proven Tensor Core support
// Simpler API, better H100 compatibility

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <cmath>
#include <iostream>

// Test if cuBLAS provides better Tensor Core algorithms
extern "C" void test_cublas_vs_cublaslt(int M, int N, int K) {
    // Allocate test matrices
    __half *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(__half));
    hipMalloc(&d_B, K * N * sizeof(__half));
    hipMalloc(&d_C, M * N * sizeof(__half));
    
    // Fill with random data
    // (skipped for brevity)
    
    // cuBLAS approach
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);  // Force Tensor Cores!
    
    __half alpha = __float2half(1.0f);
    __half beta = __float2half(0.0f);
    
    // Warmup
    for (int i = 0; i < 10; i++) {
        hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                    M, N, K,
                    &alpha,
                    d_A, M,
                    d_B, N,
                    &beta,
                    d_C, M);
    }
    
    // Benchmark
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    for (int i = 0; i < 100; i++) {
        hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                    M, N, K,
                    &alpha,
                    d_A, M,
                    d_B, N,
                    &beta,
                    d_C, M);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    ms /= 100.0f;  // Average
    
    // Compute TFLOPS
    double flops = 2.0 * M * N * K;
    double tflops = (flops / (ms * 1e-3)) / 1e12;
    
    std::cout << "cuBLAS Hgemm: " << M << "×" << K << " @ " << K << "×" << N
              << " = " << ms << " ms (" << tflops << " TFLOPS)\n";
    
    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);
}

