#include "hip/hip_runtime.h"
/**
 * Child-V2c-v6a: WMMA GREEN Fix (store → softmax → rebuild)
 * 
 * FIX from v6: WMMA fragment elements don't map linearly to (row, col)
 * SOLUTION: Store → operate row-wise → rebuild → load
 * 
 * APPROACH:
 * 1. Store Q@K^T fragment to per-warp SMEM scratch (16×16 float)
 * 2. Row-wise streaming softmax on stored scores
 * 3. Build P fragment (16×16 half) from softmax results
 * 4. Load P fragment and WMMA P@V
 * 
 * SMEM: Added 2 small per-warp buffers (~2 KB total)
 * TARGET: 100% correctness (GREEN)
 */

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <cstdint>
#include <cfloat>
#include <cstdio>
#include "runtime.hpp"
#include "nvtx.hpp"

using namespace nvcuda;

// Tile config
template<int HEAD_DIM>
struct TileConfig {
    static constexpr int M = 64;
    static constexpr int N = (HEAD_DIM == 64) ? 64 : 32;
    static constexpr int K = HEAD_DIM;
};

#define NUM_WARPS 8
#define THREADS_PER_BLOCK (NUM_WARPS * 32)
#define HEAD_DIM_PAD(d) ((d) + 8)

// WMMA tile sizes
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// Legal cp.async helper
__device__ __forceinline__ void cp_async_vec(void* smem, const void* gmem, int bytes) {
    unsigned sm = __cvta_generic_to_shared(smem);
    if ((bytes == 16) && (((uintptr_t)smem & 0xF) == 0) && (((uintptr_t)gmem & 0xF) == 0)) {
        asm volatile("cp.async.cg.shared.global [%0], [%1], 16;" :: "r"(sm), "l"(gmem));
    } else if ((bytes == 8) && (((uintptr_t)smem & 0x7) == 0) && (((uintptr_t)gmem & 0x7) == 0)) {
        asm volatile("cp.async.cg.shared.global [%0], [%1], 8;"  :: "r"(sm), "l"(gmem));
    } else {
        asm volatile("cp.async.cg.shared.global [%0], [%1], 4;"  :: "r"(sm), "l"(gmem));
    }
}

__device__ __forceinline__ void cp_async_commit_group() {
    asm volatile("cp.async.commit_group;\n" ::);
}

template<int N>
__device__ __forceinline__ void cp_async_wait_group() {
    asm volatile("cp.async.wait_group %0;\n" :: "n"(N));
}

/**
 * SMEM Layout for V2c-v6a (with per-warp scratch for store→softmax→rebuild)
 */
template<int HEAD_DIM, int STAGES>
struct SmemLayout {
    static constexpr int M = TileConfig<HEAD_DIM>::M;
    static constexpr int N = TileConfig<HEAD_DIM>::N;
    static constexpr int Dpad = HEAD_DIM_PAD(HEAD_DIM);
    
    // Per-warp scratch (INSIGHT: per_warp_scratch)
    static constexpr int COMPUTE_WARPS = M / WMMA_M;  // e.g., 64/16 = 4
    
    // SMEM sizes
    static constexpr size_t sQ_bytes = M * Dpad * sizeof(half);
    static constexpr size_t sK_bytes = Dpad * STAGES * N * sizeof(half);
    static constexpr size_t sV_bytes = STAGES * N * Dpad * sizeof(half);
    static constexpr size_t O_accum_bytes = M * Dpad * sizeof(float);
    static constexpr size_t m_bytes = M * sizeof(float);
    static constexpr size_t l_bytes = M * sizeof(float);
    
    // Per-warp scratch: scores (float) and probs (half) for 16×16 tiles
    static constexpr size_t sS_frag_bytes = COMPUTE_WARPS * WMMA_M * WMMA_N * sizeof(float);
    static constexpr size_t sP_frag_bytes = COMPUTE_WARPS * WMMA_M * WMMA_N * sizeof(half);
    
    static constexpr size_t total_bytes = 
        sQ_bytes + sK_bytes + sV_bytes + O_accum_bytes + m_bytes + l_bytes 
        + sS_frag_bytes + sP_frag_bytes;
    
    __device__ static void get_pointers(
        char* base,
        half*& sQ, half*& sK, half*& sV,
        float*& O_accum, float*& m_smem, float*& l_smem,
        float*& sS_all, half*& sP_all
    ) {
        size_t offset = 0;
        sQ = reinterpret_cast<half*>(base + offset); offset += sQ_bytes;
        sK = reinterpret_cast<half*>(base + offset); offset += sK_bytes;
        sV = reinterpret_cast<half*>(base + offset); offset += sV_bytes;
        O_accum = reinterpret_cast<float*>(base + offset); offset += O_accum_bytes;
        m_smem = reinterpret_cast<float*>(base + offset); offset += m_bytes;
        l_smem = reinterpret_cast<float*>(base + offset); offset += l_bytes;
        // INSIGHT: per-warp scratch
        sS_all = reinterpret_cast<float*>(base + offset); offset += sS_frag_bytes;
        sP_all = reinterpret_cast<half*>(base + offset);
    }
};

/**
 * V2c-v6a Kernel: WMMA GREEN (store → softmax → rebuild)
 */
template<typename T, int HEAD_DIM, int STAGES>
__global__ void __launch_bounds__(THREADS_PER_BLOCK, 2)
sdpa_fused_v2c_v6a_kernel(
    const T* Q, const T* K, const T* V, T* O,
    int B, int H, int L, int d,
    float scale, bool causal
) {
    constexpr int M = TileConfig<HEAD_DIM>::M;
    constexpr int N = TileConfig<HEAD_DIM>::N;
    constexpr int Dpad = HEAD_DIM_PAD(HEAD_DIM);
    using Layout = SmemLayout<HEAD_DIM, STAGES>;
    
    const int tid = threadIdx.x;
    const int warp_id = tid / 32;
    const int lane = tid % 32;
    
    const int bh = blockIdx.y;
    const int q_block = blockIdx.x;
    const int q_start = q_block * M;
    const int q_end = min(q_start + M, L);
    const int num_q_rows = q_end - q_start;
    if (num_q_rows <= 0) return;
    
    // Global memory base
    const T* Q_bh = Q + bh * L * d;
    const T* K_bh = K + bh * L * d;
    const T* V_bh = V + bh * L * d;
    T* O_bh = O + bh * L * d;
    
    // Dynamic SMEM
    extern __shared__ char smem_base[];
    half *sQ, *sK, *sV;
    float *O_accum, *m_smem, *l_smem;
    float *sS_all;
    half *sP_all;
    Layout::get_pointers(smem_base, sQ, sK, sV, O_accum, m_smem, l_smem, sS_all, sP_all);
    
    // Compute warp mapping
    const int compute_warps = M / WMMA_M;
    const bool is_compute_warp = (warp_id < compute_warps);
    const int warp_tile_id = warp_id;
    
    // Per-warp scratch slices
    float* sS_frag = sS_all + warp_tile_id * (WMMA_M * WMMA_N);
    half* sP_frag = sP_all + warp_tile_id * (WMMA_M * WMMA_N);
    
    // Each compute warp owns exactly one 16-row stripe
    const int warp_m0 = is_compute_warp ? (warp_tile_id * WMMA_M) : 0;
    
    if (is_compute_warp && (warp_m0 + WMMA_M > num_q_rows)) {
        return;  // Skip partial tiles
    }
    
    // Load Q tile
    for (int idx = tid; idx < num_q_rows * HEAD_DIM; idx += blockDim.x) {
        int r = idx / HEAD_DIM;
        int c = idx % HEAD_DIM;
        if (q_start + r < L) {
            sQ[r * Dpad + c] = __ldg(&Q_bh[(q_start + r) * d + c]);
        } else {
            sQ[r * Dpad + c] = __float2half(0.0f);
        }
    }
    
    // Initialize stats
    if (is_compute_warp) {
        for (int r = 0; r < WMMA_M; ++r) {
            if (lane == 0) {
                m_smem[warp_m0 + r] = -FLT_MAX;
                l_smem[warp_m0 + r] = 0.0f;
            }
        }
    }
    
    // Initialize O_accum
    for (int idx = tid; idx < num_q_rows * HEAD_DIM; idx += blockDim.x) {
        O_accum[idx] = 0.0f;
    }
    __syncthreads();
    
    const int num_kv_tiles = (L + N - 1) / N;
    int stage = 0;
    
    // Main loop over K/V tiles
    for (int t = 0; t < num_kv_tiles; ++t) {
        const int kv_start = t * N;
        const int kv_end = min(kv_start + N, L);
        const int kv_len = kv_end - kv_start;
        
        const int read_stage = stage;
        
        // Load K transposed and V
        for (int idx = tid; idx < kv_len * HEAD_DIM; idx += blockDim.x) {
            int n = idx / HEAD_DIM;
            int c = idx % HEAD_DIM;
            
            if (kv_start + n < L) {
                // K: col-major (FIX: correct indexing)
                int k_col = read_stage * N + n;
                int k_row = c;
                int k_idx = k_col * Dpad + k_row;
                sK[k_idx] = __ldg(&K_bh[(kv_start + n) * d + c]);
                
                // V: row-major
                int v_idx = (read_stage * N + n) * Dpad + c;
                sV[v_idx] = __ldg(&V_bh[(kv_start + n) * d + c]);
            } else {
                int k_idx = (read_stage * N + n) * Dpad + c;
                sK[k_idx] = __float2half(0.0f);
                int v_idx = (read_stage * N + n) * Dpad + c;
                sV[v_idx] = __float2half(0.0f);
            }
        }
        __syncthreads();
        
        // WMMA pipeline: Q@K^T + store → softmax → rebuild → P@V
        if (is_compute_warp) {
            // Process warp's 16 rows across N columns in 16-col tiles
            for (int n0 = 0; n0 < kv_len; n0 += WMMA_N) {
                int n_tile_cols = min(WMMA_N, kv_len - n0);
                
                // WMMA Q@K^T
                wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> qk_frag;
                wmma::fill_fragment(qk_frag, 0.0f);
                
                for (int k0 = 0; k0 < HEAD_DIM; k0 += WMMA_K) {
                    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> q_frag;
                    const half* q_ptr = &sQ[warp_m0 * Dpad + k0];
                    wmma::load_matrix_sync(q_frag, q_ptr, Dpad);
                    
                    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> kt_frag;
                    const half* kt_ptr = &sK[(read_stage * N + n0) * Dpad + k0];
                    wmma::load_matrix_sync(kt_frag, kt_ptr, Dpad);  // FIX: ld = Dpad
                    
                    wmma::mma_sync(qk_frag, q_frag, kt_frag, qk_frag);
                }
                
                // Scale scores
                #pragma unroll
                for (int i = 0; i < qk_frag.num_elements; ++i) {
                    qk_frag.x[i] *= scale;
                }
                
                // FIX: wmma_store_softmax_rebuild
                // Store fragment to SMEM for row-wise processing
                wmma::store_matrix_sync(sS_frag, qk_frag, WMMA_N, wmma::mem_row_major);
                
                // Row-wise streaming softmax on this warp's 16 rows
                for (int r_local = 0; r_local < WMMA_M; ++r_local) {
                    int r_global = warp_m0 + r_local;
                    
                    // Find max and apply causal mask
                    float row_max = -FLT_MAX;
                    #pragma unroll
                    for (int n_local = 0; n_local < WMMA_N; ++n_local) {
                        float score = sS_frag[r_local * WMMA_N + n_local];
                        
                        // Apply causal mask
                        if (causal) {
                            int q_pos = q_start + r_global;
                            int k_pos = kv_start + n0 + n_local;
                            if (k_pos > q_pos) score = -FLT_MAX;
                        }
                        
                        sS_frag[r_local * WMMA_N + n_local] = score;
                        row_max = fmaxf(row_max, score);
                    }
                    
                    // Update global max
                    float m_old = m_smem[r_global];
                    float m_new = fmaxf(m_old, row_max);
                    
                    // Compute exp and sum for this tile
                    float tile_sum = 0.0f;
                    #pragma unroll
                    for (int n_local = 0; n_local < WMMA_N; ++n_local) {
                        float p = __expf(sS_frag[r_local * WMMA_N + n_local] - m_new);
                        tile_sum += p;
                        sP_frag[r_local * WMMA_N + n_local] = __float2half(p);
                    }
                    
                    // Update global sum with rescaling
                    float l_old = l_smem[r_global];
                    float rescale = __expf(m_old - m_new);
                    float l_new = l_old * rescale + tile_sum;
                    
                    // Rescale O_accum for this row
                    for (int c = lane; c < HEAD_DIM; c += 32) {
                        O_accum[r_global * Dpad + c] *= rescale;
                    }
                    
                    // Update stats
                    if (lane == 0) {
                        m_smem[r_global] = m_new;
                        l_smem[r_global] = l_new;
                    }
                }
                
                __syncwarp();
                
                // INSIGHT: pv_wmma_row_major
                // Load rebuilt P fragment for WMMA P@V
                wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> p_frag;
                wmma::load_matrix_sync(p_frag, sP_frag, WMMA_N);
                
                // WMMA P@V: accumulate into O
                for (int d0 = 0; d0 < HEAD_DIM; d0 += WMMA_K) {
                    // Load V[n0:n0+16, d0:d0+16] row-major
                    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> v_frag;
                    const half* v_ptr = &sV[(read_stage * N + n0) * Dpad + d0];
                    wmma::load_matrix_sync(v_frag, v_ptr, Dpad);
                    
                    // Load current O_accum for this tile
                    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> o_frag;
                    float* o_ptr = &O_accum[warp_m0 * Dpad + d0];
                    wmma::load_matrix_sync(o_frag, o_ptr, Dpad, wmma::mem_row_major);
                    
                    // MMA: O += P @ V
                    wmma::mma_sync(o_frag, p_frag, v_frag, o_frag);
                    
                    // Store back to O_accum
                    wmma::store_matrix_sync(o_ptr, o_frag, Dpad, wmma::mem_row_major);
                }
            }
        }
        
        __syncthreads();
        stage = (stage + 1) % STAGES;
    }
    
    // Epilogue
    for (int idx = tid; idx < num_q_rows * HEAD_DIM; idx += blockDim.x) {
        int r = idx / HEAD_DIM;
        int c = idx % HEAD_DIM;
        if (r < num_q_rows) {
            float o_val = O_accum[r * Dpad + c] / l_smem[r];
            O_bh[(q_start + r) * d + c] = __float2half(o_val);
        }
    }
}

// Explicit instantiations
template __global__ void sdpa_fused_v2c_v6a_kernel<half, 64, 2>(
    const half*, const half*, const half*, half*, int, int, int, int, float, bool);
template __global__ void sdpa_fused_v2c_v6a_kernel<half, 128, 2>(
    const half*, const half*, const half*, half*, int, int, int, int, float, bool);
template __global__ void sdpa_fused_v2c_v6a_kernel<half, 64, 3>(
    const half*, const half*, const half*, half*, int, int, int, int, float, bool);
template __global__ void sdpa_fused_v2c_v6a_kernel<half, 128, 3>(
    const half*, const half*, const half*, half*, int, int, int, int, float, bool);

// Runtime dispatcher
hipError_t sdpa_fused_forward_v2c_v6a(const SdpaParams& params, hipStream_t stream) {
    const int M = (params.d == 64) ? TileConfig<64>::M : TileConfig<128>::M;
    const int STAGES = (params.d == 128) ? 2 : ((params.L >= 2048) ? 3 : 2);
    
    dim3 grid((params.L + M - 1) / M, params.B * params.H);
    dim3 block(THREADS_PER_BLOCK);
    
    size_t smem_bytes;
    if (params.d == 64) {
        smem_bytes = (STAGES == 2) ? 
            SmemLayout<64, 2>::total_bytes : SmemLayout<64, 3>::total_bytes;
    } else {
        smem_bytes = (STAGES == 2) ?
            SmemLayout<128, 2>::total_bytes : SmemLayout<128, 3>::total_bytes;
    }
    
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    
    size_t max_smem = prop.sharedMemPerBlockOptin;
    if (smem_bytes > max_smem) {
        printf("[V2c-v6a WARNING] SMEM %zu KB > limit %zu KB\n", smem_bytes / 1024, max_smem / 1024);
        return hipErrorInvalidConfiguration;
    }
    
    static bool first_launch = true;
    if (first_launch) {
        printf("[V2c-v6a GREEN] d=%d, L=%d, M=%d, N=%d, STAGES=%d, SMEM=%zu KB (per-warp scratch)\n",
               params.d, params.L, M, 
               (params.d == 64) ? TileConfig<64>::N : TileConfig<128>::N,
               STAGES, smem_bytes / 1024);
        first_launch = false;
    }
    
    hipError_t err = hipSuccess;
    
    if (params.d == 64 && STAGES == 2) {
        auto kernel_func = sdpa_fused_v2c_v6a_kernel<half, 64, 2>;
        err = hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_func), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
        if (err != hipSuccess) return err;
        kernel_func<<<grid, block, smem_bytes, stream>>>(
            reinterpret_cast<const half*>(params.Q), reinterpret_cast<const half*>(params.K),
            reinterpret_cast<const half*>(params.V), reinterpret_cast<half*>(params.O),
            params.B, params.H, params.L, params.d, params.scale, params.causal
        );
    } else if (params.d == 64 && STAGES == 3) {
        auto kernel_func = sdpa_fused_v2c_v6a_kernel<half, 64, 3>;
        err = hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_func), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
        if (err != hipSuccess) return err;
        kernel_func<<<grid, block, smem_bytes, stream>>>(
            reinterpret_cast<const half*>(params.Q), reinterpret_cast<const half*>(params.K),
            reinterpret_cast<const half*>(params.V), reinterpret_cast<half*>(params.O),
            params.B, params.H, params.L, params.d, params.scale, params.causal
        );
    } else if (params.d == 128 && STAGES == 2) {
        auto kernel_func = sdpa_fused_v2c_v6a_kernel<half, 128, 2>;
        err = hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_func), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
        if (err != hipSuccess) return err;
        kernel_func<<<grid, block, smem_bytes, stream>>>(
            reinterpret_cast<const half*>(params.Q), reinterpret_cast<const half*>(params.K),
            reinterpret_cast<const half*>(params.V), reinterpret_cast<half*>(params.O),
            params.B, params.H, params.L, params.d, params.scale, params.causal
        );
    } else {
        auto kernel_func = sdpa_fused_v2c_v6a_kernel<half, 128, 3>;
        err = hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_func), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
        if (err != hipSuccess) return err;
        kernel_func<<<grid, block, smem_bytes, stream>>>(
            reinterpret_cast<const half*>(params.Q), reinterpret_cast<const half*>(params.K),
            reinterpret_cast<const half*>(params.V), reinterpret_cast<half*>(params.O),
            params.B, params.H, params.L, params.d, params.scale, params.causal
        );
    }
    
    return hipGetLastError();
}

