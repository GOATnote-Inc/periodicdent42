// flashcore_fa3_v4.cu
// FA-3 v4: CORRECT loop order (K/V outer, Q inner)
// This is the TRUE FlashAttention-3 architecture!
// Expected: ~60 μs (100× faster than v3.1!)

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <math.h>

#ifndef WARPS_PER_BLOCK
#define WARPS_PER_BLOCK 4
#endif
#ifndef M_TILE
#define M_TILE 64
#endif
#ifndef N_TILE
#define N_TILE 64
#endif
#ifndef PAD
#define PAD 8
#endif

__device__ __forceinline__ float warp_sum(float v) {
  #pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) {
    v += __shfl_xor_sync(0xffffffff, v, mask);
  }
  return v;
}

__global__ void __launch_bounds__(WARPS_PER_BLOCK * 32, 2)
flash3_fused_attention_fp16_kernel_v4(
    const half* __restrict__ Q,
    const half* __restrict__ K,
    const half* __restrict__ V,
    half* __restrict__ O,
    int B, int H, int S, int D,
    int is_causal,
    float scale)
{
  const int warp_id = threadIdx.x / 32;
  const int lane_id = threadIdx.x % 32;
  const int nwarps = WARPS_PER_BLOCK;

  const int q_tile_start = blockIdx.x * M_TILE;
  const int h = blockIdx.y;
  const int b = blockIdx.z;

  if (h >= H || b >= B) return;

  const int q_tile_len = min(M_TILE, S - q_tile_start);
  if (q_tile_len <= 0) return;

  const size_t base = ((size_t)b * H + (size_t)h) * (size_t)S * (size_t)D;

  const half* __restrict__ Q_bh = Q + base;
  const half* __restrict__ K_bh = K + base;
  const half* __restrict__ V_bh = V + base;
  half*       __restrict__ O_bh = O + base;

  // Shared memory with padding
  const int LDQ = D + PAD;
  const int LDK = D + PAD;
  const int LDV = D + PAD;

  extern __shared__ half smem[];
  half* smem_Q = smem;
  half* smem_K = smem_Q + (size_t)M_TILE * LDQ;
  half* smem_V = smem_K + (size_t)N_TILE * LDK;

  // Load Q tile ONCE for entire block
  {
    const int q_elems = q_tile_len * D;
    for (int idx = threadIdx.x; idx < q_elems; idx += blockDim.x) {
      const int r = idx / D;
      const int d = idx % D;
      smem_Q[r * LDQ + d] = Q_bh[(q_tile_start + r) * D + d];
    }
  }
  __syncthreads();

  // Each lane processes D/32 elements
  const int elems_per_lane = D / 32;
  
  // === REGISTER STATE: Each warp maintains state for ALL its rows ===
  // Warp owns rows: warp_id, warp_id + nwarps, warp_id + 2*nwarps, ...
  // Max rows per warp: ceil(M_TILE / nwarps) = ceil(64/4) = 16
  constexpr int MAX_ROWS_PER_WARP = (M_TILE + WARPS_PER_BLOCK - 1) / WARPS_PER_BLOCK;
  
  float m_state[MAX_ROWS_PER_WARP];      // max values
  float l_state[MAX_ROWS_PER_WARP];      // sum of exp
  float out_state[MAX_ROWS_PER_WARP][4]; // output accumulators (D=64 → 4 chunks)
  
  // Preload Q rows into registers and initialize state
  float q_rows[MAX_ROWS_PER_WARP][4];    // Q data for this warp's rows
  int row_count = 0;
  
  for (int q_local = warp_id; q_local < q_tile_len; q_local += nwarps) {
    // Load Q row
    #pragma unroll
    for (int c = 0; c < elems_per_lane; ++c) {
      const int d_idx = c * 32 + lane_id;
      q_rows[row_count][c] = __half2float(smem_Q[q_local * LDQ + d_idx]);
    }
    
    // Initialize state
    m_state[row_count] = -INFINITY;
    l_state[row_count] = 0.0f;
    #pragma unroll
    for (int c = 0; c < elems_per_lane; ++c) {
      out_state[row_count][c] = 0.0f;
    }
    
    row_count++;
  }

  // === CORRECT ARCHITECTURE: Outer loop over K/V tiles ===
  for (int k_start = 0; k_start < S; k_start += N_TILE) {
    const int k_len = min(N_TILE, S - k_start);

    // ALL WARPS COOPERATIVELY LOAD K/V TILE (ONCE!)
    {
      const int kv_elems = k_len * D;
      for (int idx = threadIdx.x; idx < kv_elems; idx += blockDim.x) {
        const int j = idx / D;
        const int d = idx % D;
        smem_K[j * LDK + d] = K_bh[(k_start + j) * D + d];
        smem_V[j * LDV + d] = V_bh[(k_start + j) * D + d];
      }
    }
    __syncthreads();

    // === Inner loop: Each warp processes ALL its rows with this K/V tile ===
    for (int row_idx = 0; row_idx < row_count; ++row_idx) {
      const int q_local = warp_id + row_idx * nwarps;
      const int q_abs = q_tile_start + q_local;

      // Process each key in this tile
      for (int j = 0; j < k_len; ++j) {
        const int key_abs = k_start + j;

        // Compute Q·K[j] using preloaded Q
        float partial = 0.0f;
        #pragma unroll
        for (int c = 0; c < elems_per_lane; ++c) {
          const int d_idx = c * 32 + lane_id;
          float k_val = __half2float(smem_K[j * LDK + d_idx]);
          partial += q_rows[row_idx][c] * k_val;
        }
        
        float score = warp_sum(partial) * scale;

        // Causal mask
        if (is_causal && key_abs > q_abs) {
          score = -INFINITY;
        }

        // Online softmax update
        float m_new = fmaxf(m_state[row_idx], score);
        float alpha = expf(m_state[row_idx] - m_new);
        float beta = expf(score - m_new);
        float l_new = l_state[row_idx] * alpha + beta;

        // Update output accumulator
        #pragma unroll
        for (int c = 0; c < elems_per_lane; ++c) {
          const int d_idx = c * 32 + lane_id;
          float v_val = __half2float(smem_V[j * LDV + d_idx]);
          out_state[row_idx][c] = out_state[row_idx][c] * alpha + beta * v_val;
        }

        // Commit new state
        m_state[row_idx] = m_new;
        l_state[row_idx] = l_new;
      }
    }

    __syncthreads();  // Wait before loading next K/V tile
  }

  // === Write outputs: Normalize and store ===
  row_count = 0;
  for (int q_local = warp_id; q_local < q_tile_len; q_local += nwarps) {
    const int q_abs = q_tile_start + q_local;
    float l_i = l_state[row_count];
    
    #pragma unroll
    for (int c = 0; c < elems_per_lane; ++c) {
      const int d_idx = c * 32 + lane_id;
      float out_val = (l_i > 0.0f) ? (out_state[row_count][c] / l_i) : 0.0f;
      O_bh[q_abs * D + d_idx] = __float2half(out_val);
    }
    
    row_count++;
  }
}

// -------------------- Host launcher --------------------
void launch_flash3_v4(
    const half* Q, const half* K, const half* V, half* O,
    int B, int H, int S, int D,
    bool is_causal,
    hipStream_t stream)
{
  const float scale = 1.0f / sqrtf((float)D);

  dim3 block(WARPS_PER_BLOCK * 32);
  dim3 grid((S + M_TILE - 1) / M_TILE, H, B);

  // Shared memory: Q + K + V (single-buffered)
  const size_t smem_elems =
      (size_t)M_TILE * (D + PAD)   // Q
    + (size_t)N_TILE * (D + PAD)   // K
    + (size_t)N_TILE * (D + PAD);  // V
  const size_t smem_bytes = smem_elems * sizeof(half);

  hipFuncSetAttribute(
      reinterpret_cast<const void*>(flash3_fused_attention_fp16_kernel_v4),
      hipFuncAttributeMaxDynamicSharedMemorySize,
      (int)smem_bytes);

  flash3_fused_attention_fp16_kernel_v4<<<grid, block, smem_bytes, stream>>>(
      Q, K, V, O, B, H, S, D, (int)is_causal, scale);
}

