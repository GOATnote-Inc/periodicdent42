// flashcore_fa3_v6_wmma.cu
// Phase 1: WMMA for Q·K^T (Tensor Cores!)
// Expected: 2122 → 400-600 μs (4-5× speedup)
// Target: <40 μs after all phases

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <math.h>

using namespace nvcuda;

#ifndef WARPS_PER_BLOCK
#define WARPS_PER_BLOCK 4
#endif
#ifndef M_TILE
#define M_TILE 64
#endif
#ifndef N_TILE
#define N_TILE 64
#endif
#ifndef PAD
#define PAD 8
#endif

// WMMA tile dimensions
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

__device__ __forceinline__ float warp_sum(float v) {
  #pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) {
    v += __shfl_xor_sync(0xffffffff, v, mask);
  }
  return v;
}

__device__ __forceinline__ float warp_max(float v) {
  #pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) {
    v = fmaxf(v, __shfl_xor_sync(0xffffffff, v, mask));
  }
  return v;
}

__global__ void __launch_bounds__(WARPS_PER_BLOCK * 32, 2)
flash3_wmma_qk_kernel(
    const half* __restrict__ Q,
    const half* __restrict__ K,
    const half* __restrict__ V,
    half* __restrict__ O,
    int B, int H, int S, int D,
    int is_causal,
    float scale)
{
  const int warp_id = threadIdx.x / 32;
  const int lane_id = threadIdx.x % 32;

  const int q_tile_start = blockIdx.x * M_TILE;
  const int h = blockIdx.y;
  const int b = blockIdx.z;

  if (h >= H || b >= B) return;

  const int q_tile_len = min(M_TILE, S - q_tile_start);
  if (q_tile_len <= 0) return;

  const size_t base = ((size_t)b * H + (size_t)h) * (size_t)S * (size_t)D;

  const half* __restrict__ Q_bh = Q + base;
  const half* __restrict__ K_bh = K + base;
  const half* __restrict__ V_bh = V + base;
  half*       __restrict__ O_bh = O + base;

  // Shared memory layout
  const int LDQ = D + PAD;
  const int LDK = D + PAD;
  const int LDV = D + PAD;
  const int LDS = N_TILE + PAD;

  extern __shared__ char smem_bytes[];
  
  half* sQ = reinterpret_cast<half*>(smem_bytes);
  half* sK = sQ + M_TILE * LDQ;
  half* sV = sK + N_TILE * LDK;
  
  // FP32 state after half buffers
  size_t float_offset = (M_TILE * LDQ + N_TILE * LDK + N_TILE * LDV) * sizeof(half);
  float_offset = (float_offset + 3) & ~3;
  
  float* sS = reinterpret_cast<float*>(smem_bytes + float_offset);  // Scores [M_TILE][N_TILE+PAD]
  float* sM = sS + M_TILE * LDS;  // Max values [M_TILE]
  float* sL = sM + M_TILE;        // Sum exp [M_TILE]
  float* sO = sL + M_TILE;        // Output acc [M_TILE * (D+PAD)]

  // === PROLOGUE: Load Q tile ===
  for (int idx = threadIdx.x; idx < q_tile_len * D; idx += blockDim.x) {
    int m = idx / D;
    int d = idx % D;
    sQ[m * LDQ + d] = Q_bh[(q_tile_start + m) * D + d];
  }

  // Initialize state
  for (int m = threadIdx.x; m < q_tile_len; m += blockDim.x) {
    sM[m] = -INFINITY;
    sL[m] = 0.0f;
  }
  for (int idx = threadIdx.x; idx < q_tile_len * D; idx += blockDim.x) {
    int m = idx / D;
    int d = idx % D;
    sO[m * (D + PAD) + d] = 0.0f;
  }
  __syncthreads();

  // === MAIN LOOP: Process K/V tiles ===
  for (int k_start = 0; k_start < S; k_start += N_TILE) {
    const int k_len = min(N_TILE, S - k_start);

    // Load K/V tile
    for (int idx = threadIdx.x; idx < k_len * D; idx += blockDim.x) {
      int n = idx / D;
      int d = idx % D;
      sK[n * LDK + d] = K_bh[(k_start + n) * D + d];
      sV[n * LDV + d] = V_bh[(k_start + n) * D + d];
    }
    __syncthreads();

    // === WMMA Q·K^T ===
    // Each warp computes 16 rows (warp_id * 16 to warp_id * 16 + 15)
    const int warp_m_start = warp_id * WMMA_M;
    if (warp_m_start < q_tile_len) {
      const int warp_m_end = min(warp_m_start + WMMA_M, q_tile_len);

      // Process N_TILE columns in WMMA_N chunks
      for (int n_wmma = 0; n_wmma < k_len; n_wmma += WMMA_N) {
        const int n_end = min(n_wmma + WMMA_N, k_len);
        
        if (n_end - n_wmma == WMMA_N && warp_m_end - warp_m_start == WMMA_M) {
          // Full WMMA tile
          wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> q_frag;
          wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> k_frag;
          wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> s_frag;
          
          wmma::fill_fragment(s_frag, 0.0f);

          // K dimension loop (D / WMMA_K iterations)
          #pragma unroll
          for (int k_wmma = 0; k_wmma < D; k_wmma += WMMA_K) {
            // Load Q fragment [warp_m_start:warp_m_start+16, k_wmma:k_wmma+16]
            wmma::load_matrix_sync(q_frag, &sQ[warp_m_start * LDQ + k_wmma], LDQ);
            
            // Load K^T fragment [n_wmma:n_wmma+16, k_wmma:k_wmma+16]
            // K is stored row-major, but we want K^T, so load as col_major
            wmma::load_matrix_sync(k_frag, &sK[n_wmma * LDK + k_wmma], LDK);
            
            // Accumulate: S += Q @ K^T
            wmma::mma_sync(s_frag, q_frag, k_frag, s_frag);
          }

          // Store scores to shared memory
          wmma::store_matrix_sync(&sS[warp_m_start * LDS + n_wmma], s_frag, LDS, wmma::mem_row_major);
        }
      }
    }
    __syncthreads();

    // === Online Softmax (scalar for now, per row) ===
    for (int m_local = warp_id; m_local < q_tile_len; m_local += WARPS_PER_BLOCK) {
      const int m_abs = q_tile_start + m_local;
      
      // Load previous state
      float m_prev = sM[m_local];
      float l_prev = sL[m_local];

      // Find max score in this tile (warp-reduce across N_TILE)
      float m_tile = -INFINITY;
      for (int n = lane_id; n < k_len; n += 32) {
        const int k_abs = k_start + n;
        float score = sS[m_local * LDS + n] * scale;
        
        // Causal mask
        if (is_causal && k_abs > m_abs) {
          score = -INFINITY;
        }
        
        m_tile = fmaxf(m_tile, score);
      }
      m_tile = warp_max(m_tile);

      // Compute new max
      float m_new = fmaxf(m_prev, m_tile);
      float alpha = expf(m_prev - m_new);
      
      // Compute sum of exp for this tile
      float l_tile = 0.0f;
      for (int n = lane_id; n < k_len; n += 32) {
        const int k_abs = k_start + n;
        float score = sS[m_local * LDS + n] * scale;
        
        if (is_causal && k_abs > m_abs) {
          score = -INFINITY;
        }
        
        float prob = expf(score - m_new);
        l_tile += prob;
        
        // Store prob for PV (reuse sS, convert to half)
        sS[m_local * LDS + n] = prob;  // Keep as float for now
      }
      l_tile = warp_sum(l_tile);

      float l_new = l_prev * alpha + l_tile;

      // Update O accumulator (scalar PV for now)
      for (int d = lane_id; d < D; d += 32) {
        float o_val = sO[m_local * (D + PAD) + d];
        o_val *= alpha;
        
        // P @ V (scalar)
        for (int n = 0; n < k_len; ++n) {
          float prob = sS[m_local * LDS + n];
          float v_val = __half2float(sV[n * LDV + d]);
          o_val += prob * v_val;
        }
        
        sO[m_local * (D + PAD) + d] = o_val;
      }

      // Store updated state
      if (lane_id == 0) {
        sM[m_local] = m_new;
        sL[m_local] = l_new;
      }
    }
    __syncthreads();
  }

  // === EPILOGUE: Normalize and write ===
  for (int m_local = warp_id; m_local < q_tile_len; m_local += WARPS_PER_BLOCK) {
    const int m_abs = q_tile_start + m_local;
    float l = sL[m_local];

    for (int d = lane_id; d < D; d += 32) {
      float o_val = sO[m_local * (D + PAD) + d];
      o_val = (l > 0.0f) ? (o_val / l) : 0.0f;
      O_bh[m_abs * D + d] = __float2half(o_val);
    }
  }
}

// -------------------- Host launcher --------------------
void launch_flash3_v6_wmma(
    const half* Q, const half* K, const half* V, half* O,
    int B, int H, int S, int D,
    bool is_causal,
    hipStream_t stream)
{
  const float scale = 1.0f / sqrtf((float)D);

  dim3 block(WARPS_PER_BLOCK * 32);
  dim3 grid((S + M_TILE - 1) / M_TILE, H, B);

  // Shared memory: Q + K + V (half) + S + M + L + O (float)
  const size_t half_elems = M_TILE * (D + PAD) + N_TILE * (D + PAD) + N_TILE * (D + PAD);
  const size_t float_elems = M_TILE * (N_TILE + PAD) + M_TILE + M_TILE + M_TILE * (D + PAD);
  const size_t smem_bytes = half_elems * sizeof(half) + float_elems * sizeof(float) + 4;

  hipFuncSetAttribute(
      reinterpret_cast<const void*>(flash3_wmma_qk_kernel),
      hipFuncAttributeMaxDynamicSharedMemorySize,
      (int)smem_bytes);

  flash3_wmma_qk_kernel<<<grid, block, smem_bytes, stream>>>(
      Q, K, V, O, B, H, S, D, (int)is_causal, scale);
}

