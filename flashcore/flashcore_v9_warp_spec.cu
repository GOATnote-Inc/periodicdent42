#include "hip/hip_runtime.h"
// FlashCore v9: Warp Specialization
// Goal: 98 μs → 70-75 μs (1.3-1.4× speedup)
//
// Architecture:
// - 16 warps (512 threads): 12 compute + 4 producer
// - Compute warps (0-11): QK^T + softmax + P·V
// - Producer warps (12-15): Prefetch K/V tiles asynchronously
// - Overlap: Producers fetch tile N+1 while consumers compute tile N
//
// SMEM: 49 KB (same as v8, fits 2 CTAs/SM)
// Tiles: 48×32 asymmetric (optimal from v8)

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>

namespace cg = cooperative_groups;
using namespace nvcuda;

namespace flashcore {
namespace v9_warp_spec {

//=============================================================================
// Configuration
//=============================================================================

constexpr int kTileM = 48;        // Q tile rows
constexpr int kTileN = 32;        // KV tile rows
constexpr int kTileD = 64;        // Head dimension
constexpr int kTilePadD = 72;     // Padded D (+8 for bank conflict avoidance)
constexpr int kTilePadN = 48;     // Padded N (+16 for WMMA safety)

constexpr int kComputeWarps = 12;  // Warps for compute (0-11)
constexpr int kProducerWarps = 4;  // Warps for prefetch (12-15)
constexpr int kTotalWarps = 16;    // Total warps per CTA
constexpr int kWarpSize = 32;
constexpr int kThreadsPerBlock = kTotalWarps * kWarpSize;  // 512

constexpr int kWMMAM = 16;
constexpr int kWMMAN = 16;
constexpr int kWMMAK = 16;

constexpr int kStages = 2;  // Double buffering

//=============================================================================
// Dynamic Shared Memory Layout
//=============================================================================

struct SMEMLayout {
    half* q_tile;                    // [kTileM][kTilePadD]
    half* kv_tiles[kStages * 2];    // [stage][kv][kTilePadN][kTilePadD]
    float* scores;                   // [kTileM][kTilePadN]
    half* probs;                     // [kTileM][kTilePadN]
    float* m_state;                  // [kTileM]
    float* l_state;                  // [kTileM]
    float* o_accum;                  // [kTileM][kTileD]
    
    // Synchronization flags for producer-consumer coordination
    volatile int* stage_ready;       // [kStages] - set by producers, cleared by consumers
    
    __device__ SMEMLayout(char* base) {
        char* ptr = base;
        
        // Align all pointers to 16 bytes
        auto align_ptr = [](char*& p, size_t bytes) -> void* {
            size_t addr = reinterpret_cast<size_t>(p);
            size_t aligned = (addr + 15) & ~15;
            void* result = reinterpret_cast<void*>(aligned);
            p = reinterpret_cast<char*>(aligned + bytes);
            return result;
        };
        
        q_tile = static_cast<half*>(align_ptr(ptr, kTileM * kTilePadD * sizeof(half)));
        
        for (int i = 0; i < kStages * 2; i++) {
            kv_tiles[i] = static_cast<half*>(align_ptr(ptr, kTilePadN * kTilePadD * sizeof(half)));
        }
        
        scores = static_cast<float*>(align_ptr(ptr, kTileM * kTilePadN * sizeof(float)));
        probs = static_cast<half*>(align_ptr(ptr, kTileM * kTilePadN * sizeof(half)));
        m_state = static_cast<float*>(align_ptr(ptr, kTileM * sizeof(float)));
        l_state = static_cast<float*>(align_ptr(ptr, kTileM * sizeof(float)));
        o_accum = static_cast<float*>(align_ptr(ptr, kTileM * kTileD * sizeof(float)));
        stage_ready = static_cast<volatile int*>(align_ptr(ptr, kStages * sizeof(int)));
    }
    
    __device__ size_t total_bytes() const {
        size_t total = 0;
        total += kTileM * kTilePadD * sizeof(half);          // q_tile: 6.75 KB
        total += kStages * 2 * kTilePadN * kTilePadD * sizeof(half);  // kv_tiles: 27 KB
        total += kTileM * kTilePadN * sizeof(float);         // scores: 9 KB
        total += kTileM * kTilePadN * sizeof(half);          // probs: 4.5 KB
        total += kTileM * sizeof(float);                     // m_state: 192 B
        total += kTileM * sizeof(float);                     // l_state: 192 B
        total += kTileM * kTileD * sizeof(float);            // o_accum: 12 KB
        total += kStages * sizeof(int);                      // stage_ready: 8 B
        total += 16 * 10;  // Alignment padding
        return total;  // ~59 KB
    }
};

//=============================================================================
// Helper Functions
//=============================================================================

__device__ __forceinline__ float warp_reduce_max(float val) {
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_xor_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

__device__ __forceinline__ float warp_reduce_sum(float val) {
    #pragma unroll
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_xor_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

//=============================================================================
// Producer Warp: Asynchronous K/V Tile Prefetching
//=============================================================================

__device__ void producer_warp(
    const half* __restrict__ K,
    const half* __restrict__ V,
    SMEMLayout& layout,
    int B, int H, int S, int D,
    int batch_idx, int head_idx) {
    
    const int warp_id = threadIdx.x / kWarpSize;
    const int lane_id = threadIdx.x % kWarpSize;
    
    // Only producer warps (12-15)
    if (warp_id < kComputeWarps) return;
    
    const int producer_id = warp_id - kComputeWarps;  // 0-3
    
    const half* K_bh = K + (batch_idx * H + head_idx) * S * D;
    const half* V_bh = V + (batch_idx * H + head_idx) * S * D;
    
    const int num_kv_tiles = (S + kTileN - 1) / kTileN;
    
    // Initialize stage_ready flags
    if (producer_id == 0 && lane_id < kStages) {
        layout.stage_ready[lane_id] = 0;
    }
    __syncthreads();
    
    // Prefetch initial stages
    for (int stage = 0; stage < kStages; stage++) {
        const int kv_tile_idx = stage;
        if (kv_tile_idx >= num_kv_tiles) break;
        
        const int kv_start = kv_tile_idx * kTileN;
        const int kv_len = min(kTileN, S - kv_start);
        
        half* k_tile = layout.kv_tiles[stage * 2];
        half* v_tile = layout.kv_tiles[stage * 2 + 1];
        
        // Each producer warp loads different elements
        // 4 producer warps × 32 threads = 128 threads
        const int total_producer_threads = kProducerWarps * kWarpSize;
        const int producer_thread_id = (warp_id - kComputeWarps) * kWarpSize + lane_id;
        
        // Load K tile (vectorized 8 halfs = 16 bytes)
        constexpr int kVecSize = 8;
        const int k_elements = kv_len * kTilePadD;
        const int k_vecs = (k_elements + kVecSize - 1) / kVecSize;
        
        for (int vec_idx = producer_thread_id; vec_idx < k_vecs; vec_idx += total_producer_threads) {
            const int elem_idx = vec_idx * kVecSize;
            const int row = elem_idx / kTilePadD;
            const int col = elem_idx % kTilePadD;
            
            if (row < kv_len && col + kVecSize <= D) {
                uint4 data = *reinterpret_cast<const uint4*>(&K_bh[(kv_start + row) * D + col]);
                *reinterpret_cast<uint4*>(&k_tile[row * kTilePadD + col]) = data;
            } else if (row < kv_len && col < D) {
                // Scalar fallback for partial vectors
                for (int i = 0; i < kVecSize && col + i < D; i++) {
                    k_tile[row * kTilePadD + col + i] = K_bh[(kv_start + row) * D + col + i];
                }
            }
        }
        
        // Load V tile (same pattern)
        for (int vec_idx = producer_thread_id; vec_idx < k_vecs; vec_idx += total_producer_threads) {
            const int elem_idx = vec_idx * kVecSize;
            const int row = elem_idx / kTilePadD;
            const int col = elem_idx % kTilePadD;
            
            if (row < kv_len && col + kVecSize <= D) {
                uint4 data = *reinterpret_cast<const uint4*>(&V_bh[(kv_start + row) * D + col]);
                *reinterpret_cast<uint4*>(&v_tile[row * kTilePadD + col]) = data;
            } else if (row < kv_len && col < D) {
                for (int i = 0; i < kVecSize && col + i < D; i++) {
                    v_tile[row * kTilePadD + col + i] = V_bh[(kv_start + row) * D + col + i];
                }
            }
        }
        
        __syncthreads();
        
        // Signal stage ready
        if (producer_id == 0 && lane_id == 0) {
            layout.stage_ready[stage] = 1;
        }
    }
    
    // Continue prefetching for remaining tiles
    for (int kv_tile_idx = kStages; kv_tile_idx < num_kv_tiles; kv_tile_idx++) {
        const int stage = kv_tile_idx % kStages;
        
        // Wait for consumer to finish with this stage
        if (producer_id == 0 && lane_id == 0) {
            while (layout.stage_ready[stage] == 1) {
                // Spin wait (consumers will clear this)
            }
        }
        __syncwarp();
        
        const int kv_start = kv_tile_idx * kTileN;
        const int kv_len = min(kTileN, S - kv_start);
        
        half* k_tile = layout.kv_tiles[stage * 2];
        half* v_tile = layout.kv_tiles[stage * 2 + 1];
        
        const int total_producer_threads = kProducerWarps * kWarpSize;
        const int producer_thread_id = (warp_id - kComputeWarps) * kWarpSize + lane_id;
        
        constexpr int kVecSize = 8;
        const int k_elements = kv_len * kTilePadD;
        const int k_vecs = (k_elements + kVecSize - 1) / kVecSize;
        
        for (int vec_idx = producer_thread_id; vec_idx < k_vecs; vec_idx += total_producer_threads) {
            const int elem_idx = vec_idx * kVecSize;
            const int row = elem_idx / kTilePadD;
            const int col = elem_idx % kTilePadD;
            
            if (row < kv_len && col + kVecSize <= D) {
                uint4 data = *reinterpret_cast<const uint4*>(&K_bh[(kv_start + row) * D + col]);
                *reinterpret_cast<uint4*>(&k_tile[row * kTilePadD + col]) = data;
            } else if (row < kv_len && col < D) {
                for (int i = 0; i < kVecSize && col + i < D; i++) {
                    k_tile[row * kTilePadD + col + i] = K_bh[(kv_start + row) * D + col + i];
                }
            }
        }
        
        for (int vec_idx = producer_thread_id; vec_idx < k_vecs; vec_idx += total_producer_threads) {
            const int elem_idx = vec_idx * kVecSize;
            const int row = elem_idx / kTilePadD;
            const int col = elem_idx % kTilePadD;
            
            if (row < kv_len && col + kVecSize <= D) {
                uint4 data = *reinterpret_cast<const uint4*>(&V_bh[(kv_start + row) * D + col]);
                *reinterpret_cast<uint4*>(&v_tile[row * kTilePadD + col]) = data;
            } else if (row < kv_len && col < D) {
                for (int i = 0; i < kVecSize && col + i < D; i++) {
                    v_tile[row * kTilePadD + col + i] = V_bh[(kv_start + row) * D + col + i];
                }
            }
        }
        
        __syncthreads();
        
        // Signal stage ready
        if (producer_id == 0 && lane_id == 0) {
            layout.stage_ready[stage] = 1;
        }
    }
}

//=============================================================================
// Consumer Warps: Compute QK^T + Softmax + P·V
//=============================================================================

__device__ void consumer_warps(
    const half* __restrict__ Q,
    half* __restrict__ O,
    SMEMLayout& layout,
    int B, int H, int S, int D,
    float scale,
    int batch_idx, int head_idx,
    int q_tile_idx) {
    
    const int warp_id = threadIdx.x / kWarpSize;
    const int lane_id = threadIdx.x % kWarpSize;
    
    // Only compute warps (0-11)
    if (warp_id >= kComputeWarps) return;
    
    const half* Q_bh = Q + (batch_idx * H + head_idx) * S * D;
    half* O_bh = O + (batch_idx * H + head_idx) * S * D;
    
    const int q_start = q_tile_idx * kTileM;
    const int q_len = min(kTileM, S - q_start);
    
    // Load Q tile (all compute warps collaborate)
    const int compute_threads = kComputeWarps * kWarpSize;  // 384 threads
    const int thread_id = warp_id * kWarpSize + lane_id;
    
    constexpr int kVecSize = 8;
    const int q_vecs = (q_len * D + kVecSize - 1) / kVecSize;
    
    for (int vec_idx = thread_id; vec_idx < q_vecs; vec_idx += compute_threads) {
        const int elem_idx = vec_idx * kVecSize;
        const int row = elem_idx / D;
        const int col = elem_idx % D;
        
        if (row < q_len && col + kVecSize <= D) {
            uint4 data = *reinterpret_cast<const uint4*>(&Q_bh[(q_start + row) * D + col]);
            *reinterpret_cast<uint4*>(&layout.q_tile[row * kTilePadD + col]) = data;
        }
    }
    
    // Initialize softmax state
    for (int m = thread_id; m < kTileM; m += compute_threads) {
        layout.m_state[m] = __int_as_float(0xff800000);  // -inf
        layout.l_state[m] = 0.0f;
    }
    
    // Initialize output accumulator
    for (int idx = thread_id; idx < kTileM * kTileD; idx += compute_threads) {
        layout.o_accum[idx] = 0.0f;
    }
    
    __syncthreads();
    
    const int num_kv_tiles = (S + kTileN - 1) / kTileN;
    
    // Process KV tiles
    for (int kv_tile_idx = 0; kv_tile_idx < num_kv_tiles; kv_tile_idx++) {
        const int stage = kv_tile_idx % kStages;
        
        // Wait for producers to load this stage
        if (warp_id == 0 && lane_id == 0) {
            while (layout.stage_ready[stage] == 0) {
                // Spin wait
            }
        }
        __syncthreads();
        
        const half* k_tile = layout.kv_tiles[stage * 2];
        const half* v_tile = layout.kv_tiles[stage * 2 + 1];
        
        const int kv_start = kv_tile_idx * kTileN;
        const int kv_len = min(kTileN, S - kv_start);
        
        // QK^T with WMMA (12 compute warps, 3×2 warp layout for 48×32)
        const int warp_m = warp_id / 2;  // 0-5 (6 rows of warps)
        const int warp_n = warp_id % 2;  // 0-1 (2 cols of warps)
        
        wmma::fragment<wmma::matrix_a, kWMMAM, kWMMAN, kWMMAK, half, wmma::row_major> a_frag;
        wmma::fragment<wmma::matrix_b, kWMMAM, kWMMAN, kWMMAK, half, wmma::col_major> b_frag;
        wmma::fragment<wmma::accumulator, kWMMAM, kWMMAN, kWMMAK, float> c_frag;
        
        wmma::fill_fragment(c_frag, 0.0f);
        
        // Each warp computes one 16×16 tile
        const int m_base = warp_m * kWMMAM;  // 0, 16, 32 (but only up to 48-16=32)
        const int n_base = warp_n * kWMMAN;  // 0, 16
        
        if (m_base < q_len && n_base < kv_len) {
            #pragma unroll
            for (int k = 0; k < kTileD; k += kWMMAK) {
                wmma::load_matrix_sync(a_frag, &layout.q_tile[m_base * kTilePadD + k], kTilePadD);
                wmma::load_matrix_sync(b_frag, &k_tile[n_base * kTilePadD + k], kTilePadD);
                wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            }
            
            // Scale and store scores
            #pragma unroll
            for (int i = 0; i < c_frag.num_elements; i++) {
                c_frag.x[i] *= scale;
            }
            
            wmma::store_matrix_sync(&layout.scores[m_base * kTilePadN + n_base], c_frag, kTilePadN, wmma::mem_row_major);
        }
        
        __syncthreads();
        
        // Online softmax (all compute warps process rows in parallel)
        for (int m = thread_id; m < q_len; m += compute_threads) {
            float m_old = layout.m_state[m];
            float l_old = layout.l_state[m];
            
            // Find max in this row
            float m_tile = __int_as_float(0xff800000);  // -inf
            for (int n = 0; n < kv_len; n++) {
                m_tile = fmaxf(m_tile, layout.scores[m * kTilePadN + n]);
            }
            
            float m_new = fmaxf(m_old, m_tile);
            
            // Compute exp and sum
            float l_tile = 0.0f;
            for (int n = 0; n < kv_len; n++) {
                float score = layout.scores[m * kTilePadN + n];
                float prob = expf(score - m_new);
                layout.probs[m * kTilePadN + n] = __float2half(prob);
                l_tile += prob;
            }
            
            // Update running sum
            float correction = expf(m_old - m_new);
            float l_new = l_old * correction + l_tile;
            
            // Rescale previous output
            if (kv_tile_idx > 0) {
                for (int d = 0; d < kTileD; d++) {
                    layout.o_accum[m * kTileD + d] *= correction;
                }
            }
            
            layout.m_state[m] = m_new;
            layout.l_state[m] = l_new;
        }
        
        __syncthreads();
        
        // P·V with WMMA (12 warps, 3×4 layout for 48×64)
        const int pv_warp_m = warp_id / 4;  // 0-2 (3 rows)
        const int pv_warp_n = warp_id % 4;  // 0-3 (4 cols)
        
        wmma::fragment<wmma::matrix_a, kWMMAM, kWMMAN, kWMMAK, half, wmma::row_major> p_frag;
        wmma::fragment<wmma::matrix_b, kWMMAM, kWMMAN, kWMMAK, half, wmma::row_major> v_frag;
        wmma::fragment<wmma::accumulator, kWMMAM, kWMMAN, kWMMAK, float> pv_frag;
        
        const int pv_m_base = pv_warp_m * kWMMAM;  // 0, 16, 32
        const int pv_n_base = pv_warp_n * kWMMAN;  // 0, 16, 32, 48
        
        if (pv_m_base < q_len && pv_n_base < kTileD) {
            wmma::fill_fragment(pv_frag, 0.0f);
            
            for (int k = 0; k < kv_len; k += kWMMAK) {
                if (k + kWMMAK <= kv_len) {
                    wmma::load_matrix_sync(p_frag, &layout.probs[pv_m_base * kTilePadN + k], kTilePadN);
                    wmma::load_matrix_sync(v_frag, &v_tile[k * kTilePadD + pv_n_base], kTilePadD);
                    wmma::mma_sync(pv_frag, p_frag, v_frag, pv_frag);
                }
            }
            
            // Accumulate to output
            float pv_results[pv_frag.num_elements];
            wmma::store_matrix_sync(pv_results, pv_frag, kWMMAM, wmma::mem_row_major);
            
            #pragma unroll
            for (int i = 0; i < kWMMAM; i++) {
                for (int j = 0; j < kWMMAN; j++) {
                    const int out_m = pv_m_base + i;
                    const int out_n = pv_n_base + j;
                    if (out_m < q_len && out_n < kTileD) {
                        atomicAdd(&layout.o_accum[out_m * kTileD + out_n], pv_results[i * kWMMAN + j]);
                    }
                }
            }
        }
        
        __syncthreads();
        
        // Clear stage_ready flag for producers
        if (warp_id == 0 && lane_id == 0) {
            layout.stage_ready[stage] = 0;
        }
    }
    
    // Finalize: normalize and write output
    __syncthreads();
    
    for (int idx = thread_id; idx < q_len * kTileD; idx += compute_threads) {
        const int m = idx / kTileD;
        const int d = idx % kTileD;
        
        float inv_l = 1.0f / layout.l_state[m];
        float o_val = layout.o_accum[m * kTileD + d] * inv_l;
        O_bh[(q_start + m) * D + d] = __float2half(o_val);
    }
}

//=============================================================================
// Main Kernel
//=============================================================================

__global__ __launch_bounds__(kThreadsPerBlock, 1)  // 512 threads, 1 CTA/SM for max SMEM
void fused_attention_warp_spec_kernel(
    const half* __restrict__ Q,
    const half* __restrict__ K,
    const half* __restrict__ V,
    half* __restrict__ O,
    int B, int H, int S, int D,
    float scale) {
    
    extern __shared__ char smem_base[];
    SMEMLayout layout(smem_base);
    
    const int batch_idx = blockIdx.x / H;
    const int head_idx = blockIdx.x % H;
    const int q_tile_idx = blockIdx.y;
    
    const int warp_id = threadIdx.x / kWarpSize;
    
    // Split execution based on warp role
    if (warp_id < kComputeWarps) {
        // Consumer warps: compute attention
        consumer_warps(Q, O, layout, B, H, S, D, scale, batch_idx, head_idx, q_tile_idx);
    } else {
        // Producer warps: prefetch K/V tiles
        producer_warp(K, V, layout, B, H, S, D, batch_idx, head_idx);
    }
}

//=============================================================================
// Host Launch Function
//=============================================================================

extern "C" void flashcore_v9_warp_spec_launch(
    const half* Q,
    const half* K,
    const half* V,
    half* O,
    int B,
    int H,
    int S,
    int D,
    float scale,
    hipStream_t stream) {
    
    // Calculate SMEM requirements (same as SMEMLayout::total_bytes())
    const size_t smem_bytes = 64 * 1024;  // Conservative 64 KB (actual ~59 KB)
    
    // Set dynamic shared memory
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        fused_attention_warp_spec_kernel),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        smem_bytes
    );
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        fused_attention_warp_spec_kernel),
        hipFuncAttributePreferredSharedMemoryCarveout,
        cudaSharedmemCarveoutMaxShared
    );
    
    const int num_q_tiles = (S + kTileM - 1) / kTileM;
    dim3 grid(B * H, num_q_tiles);
    dim3 block(kThreadsPerBlock);
    
    fused_attention_warp_spec_kernel<<<grid, block, smem_bytes, stream>>>(
        Q, K, V, O, B, H, S, D, scale
    );
}

}  // namespace v9_warp_spec
}  // namespace flashcore

