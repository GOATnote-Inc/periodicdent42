// flashcore_fa3_v6_opt_vec.cu
// v6 optimized: Vectorized global loads (float4 for 8×half)
// Expected: 447 → 350-400 μs (1.2× speedup from vectorization)
// Goal: Reach <100 μs with full optimization

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <math.h>

using namespace nvcuda;

#ifndef WARPS_PER_BLOCK
#define WARPS_PER_BLOCK 4
#endif
#ifndef M_TILE
#define M_TILE 64
#endif
#ifndef N_TILE
#define N_TILE 64
#endif
#ifndef PAD
#define PAD 8
#endif

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// float4 loads 8 half values (128-bit aligned)
using half8 = float4;

__device__ __forceinline__ float warp_sum(float v) {
  #pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) {
    v += __shfl_xor_sync(0xffffffff, v, mask);
  }
  return v;
}

__device__ __forceinline__ float warp_max(float v) {
  #pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) {
    v = fmaxf(v, __shfl_xor_sync(0xffffffff, v, mask));
  }
  return v;
}

__global__ void __launch_bounds__(WARPS_PER_BLOCK * 32, 2)
flash3_wmma_qk_vectorized_kernel(
    const half* __restrict__ Q,
    const half* __restrict__ K,
    const half* __restrict__ V,
    half* __restrict__ O,
    int B, int H, int S, int D,
    int is_causal,
    float scale)
{
  const int warp_id = threadIdx.x / 32;
  const int lane_id = threadIdx.x % 32;

  const int q_tile_start = blockIdx.x * M_TILE;
  const int h = blockIdx.y;
  const int b = blockIdx.z;

  if (h >= H || b >= B) return;

  const int q_tile_len = min(M_TILE, S - q_tile_start);
  if (q_tile_len <= 0) return;

  const size_t base = ((size_t)b * H + (size_t)h) * (size_t)S * (size_t)D;

  const half* __restrict__ Q_bh = Q + base;
  const half* __restrict__ K_bh = K + base;
  const half* __restrict__ V_bh = V + base;
  half*       __restrict__ O_bh = O + base;

  const int LDQ = D + PAD;
  const int LDK = D + PAD;
  const int LDV = D + PAD;
  const int LDS = N_TILE + PAD;

  extern __shared__ char smem_bytes[];
  
  half* sQ = reinterpret_cast<half*>(smem_bytes);
  half* sK = sQ + M_TILE * LDQ;
  half* sV = sK + N_TILE * LDK;
  
  size_t float_offset = (M_TILE * LDQ + N_TILE * LDK + N_TILE * LDV) * sizeof(half);
  float_offset = (float_offset + 3) & ~3;
  
  float* sS = reinterpret_cast<float*>(smem_bytes + float_offset);
  float* sM = sS + M_TILE * LDS;
  float* sL = sM + M_TILE;
  float* sO = sL + M_TILE;

  // === PROLOGUE: Load Q with vectorization ===
  // Load Q tile using float4 (8 half values at once)
  constexpr int vec_size = 8;  // float4 = 8×half
  const int q_vec_elems = (q_tile_len * D) / vec_size;
  
  for (int idx = threadIdx.x; idx < q_vec_elems; idx += blockDim.x) {
    int linear_idx = idx * vec_size;
    int m = linear_idx / D;
    int d = linear_idx % D;
    
    // Vectorized load from global
    half8 vec_data = *reinterpret_cast<const half8*>(&Q_bh[(q_tile_start + m) * D + d]);
    
    // Store to shared (aligned)
    *reinterpret_cast<half8*>(&sQ[m * LDQ + d]) = vec_data;
  }
  
  // Handle tail elements (if D not multiple of 8)
  for (int idx = q_vec_elems * vec_size + threadIdx.x; idx < q_tile_len * D; idx += blockDim.x) {
    int m = idx / D;
    int d = idx % D;
    sQ[m * LDQ + d] = Q_bh[(q_tile_start + m) * D + d];
  }

  // Initialize state
  for (int m = threadIdx.x; m < q_tile_len; m += blockDim.x) {
    sM[m] = -INFINITY;
    sL[m] = 0.0f;
  }
  for (int idx = threadIdx.x; idx < q_tile_len * D; idx += blockDim.x) {
    int m = idx / D;
    int d = idx % D;
    sO[m * (D + PAD) + d] = 0.0f;
  }
  __syncthreads();

  // === MAIN LOOP ===
  for (int k_start = 0; k_start < S; k_start += N_TILE) {
    const int k_len = min(N_TILE, S - k_start);

    // Load K/V with vectorization
    const int kv_vec_elems = (k_len * D) / vec_size;
    
    for (int idx = threadIdx.x; idx < kv_vec_elems; idx += blockDim.x) {
      int linear_idx = idx * vec_size;
      int n = linear_idx / D;
      int d = linear_idx % D;
      
      half8 k_vec = *reinterpret_cast<const half8*>(&K_bh[(k_start + n) * D + d]);
      half8 v_vec = *reinterpret_cast<const half8*>(&V_bh[(k_start + n) * D + d]);
      
      *reinterpret_cast<half8*>(&sK[n * LDK + d]) = k_vec;
      *reinterpret_cast<half8*>(&sV[n * LDV + d]) = v_vec;
    }
    
    // Tail elements
    for (int idx = kv_vec_elems * vec_size + threadIdx.x; idx < k_len * D; idx += blockDim.x) {
      int n = idx / D;
      int d = idx % D;
      sK[n * LDK + d] = K_bh[(k_start + n) * D + d];
      sV[n * LDV + d] = V_bh[(k_start + n) * D + d];
    }
    __syncthreads();

    // === WMMA Q·K^T (same as v6) ===
    const int warp_m_start = warp_id * WMMA_M;
    if (warp_m_start < q_tile_len) {
      const int warp_m_end = min(warp_m_start + WMMA_M, q_tile_len);

      for (int n_wmma = 0; n_wmma < k_len; n_wmma += WMMA_N) {
        const int n_end = min(n_wmma + WMMA_N, k_len);
        
        if (n_end - n_wmma == WMMA_N && warp_m_end - warp_m_start == WMMA_M) {
          wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> q_frag;
          wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> k_frag;
          wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> s_frag;
          
          wmma::fill_fragment(s_frag, 0.0f);

          #pragma unroll
          for (int k_wmma = 0; k_wmma < D; k_wmma += WMMA_K) {
            wmma::load_matrix_sync(q_frag, &sQ[warp_m_start * LDQ + k_wmma], LDQ);
            wmma::load_matrix_sync(k_frag, &sK[n_wmma * LDK + k_wmma], LDK);
            wmma::mma_sync(s_frag, q_frag, k_frag, s_frag);
          }

          wmma::store_matrix_sync(&sS[warp_m_start * LDS + n_wmma], s_frag, LDS, wmma::mem_row_major);
        }
      }
    }
    __syncthreads();

    // === Online Softmax (same as v6) ===
    for (int m_local = warp_id; m_local < q_tile_len; m_local += WARPS_PER_BLOCK) {
      const int m_abs = q_tile_start + m_local;
      
      float m_prev = sM[m_local];
      float l_prev = sL[m_local];

      float m_tile = -INFINITY;
      for (int n = lane_id; n < k_len; n += 32) {
        const int k_abs = k_start + n;
        float score = sS[m_local * LDS + n] * scale;
        if (is_causal && k_abs > m_abs) score = -INFINITY;
        m_tile = fmaxf(m_tile, score);
      }
      m_tile = warp_max(m_tile);

      float m_new = fmaxf(m_prev, m_tile);
      float alpha = expf(m_prev - m_new);
      
      float l_tile = 0.0f;
      for (int n = lane_id; n < k_len; n += 32) {
        const int k_abs = k_start + n;
        float score = sS[m_local * LDS + n] * scale;
        if (is_causal && k_abs > m_abs) score = -INFINITY;
        
        float prob = expf(score - m_new);
        l_tile += prob;
        sS[m_local * LDS + n] = prob;
      }
      l_tile = warp_sum(l_tile);

      float l_new = l_prev * alpha + l_tile;

      // Scalar PV (for now)
      for (int d = lane_id; d < D; d += 32) {
        float o_val = sO[m_local * (D + PAD) + d];
        o_val *= alpha;
        
        for (int n = 0; n < k_len; ++n) {
          float prob = sS[m_local * LDS + n];
          float v_val = __half2float(sV[n * LDV + d]);
          o_val += prob * v_val;
        }
        
        sO[m_local * (D + PAD) + d] = o_val;
      }

      if (lane_id == 0) {
        sM[m_local] = m_new;
        sL[m_local] = l_new;
      }
    }
    __syncthreads();
  }

  // === EPILOGUE: Vectorized output ===
  for (int m_local = warp_id; m_local < q_tile_len; m_local += WARPS_PER_BLOCK) {
    const int m_abs = q_tile_start + m_local;
    float l = sL[m_local];

    // Vectorized stores (float4 = 8 half)
    const int d_vec_count = D / vec_size;
    
    for (int d_vec = lane_id; d_vec < d_vec_count; d_vec += 32) {
      int d_base = d_vec * vec_size;
      
      half temp[8];
      #pragma unroll
      for (int i = 0; i < 8; ++i) {
        float o_val = sO[m_local * (D + PAD) + d_base + i];
        o_val = (l > 0.0f) ? (o_val / l) : 0.0f;
        temp[i] = __float2half(o_val);
      }
      
      *reinterpret_cast<half8*>(&O_bh[m_abs * D + d_base]) = *reinterpret_cast<half8*>(temp);
    }
    
    // Tail elements
    for (int d = d_vec_count * vec_size + lane_id; d < D; d += 32) {
      float o_val = sO[m_local * (D + PAD) + d];
      o_val = (l > 0.0f) ? (o_val / l) : 0.0f;
      O_bh[m_abs * D + d] = __float2half(o_val);
    }
  }
}

void launch_flash3_v6_opt_vec(
    const half* Q, const half* K, const half* V, half* O,
    int B, int H, int S, int D,
    bool is_causal,
    hipStream_t stream)
{
  const float scale = 1.0f / sqrtf((float)D);

  dim3 block(WARPS_PER_BLOCK * 32);
  dim3 grid((S + M_TILE - 1) / M_TILE, H, B);

  const size_t half_elems = M_TILE * (D + PAD) + N_TILE * (D + PAD) + N_TILE * (D + PAD);
  const size_t float_elems = M_TILE * (N_TILE + PAD) + M_TILE + M_TILE + M_TILE * (D + PAD);
  const size_t smem_bytes = half_elems * sizeof(half) + float_elems * sizeof(float) + 4;

  hipFuncSetAttribute(
      reinterpret_cast<const void*>(flash3_wmma_qk_vectorized_kernel),
      hipFuncAttributeMaxDynamicSharedMemorySize,
      (int)smem_bytes);

  flash3_wmma_qk_vectorized_kernel<<<grid, block, smem_bytes, stream>>>(
      Q, K, V, O, B, H, S, D, (int)is_causal, scale);
}

