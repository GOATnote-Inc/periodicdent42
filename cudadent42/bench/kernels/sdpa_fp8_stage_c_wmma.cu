#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <cstdint>
#include <cmath>
#include <cstdio>
#include <cuda_pipeline_primitives.h>

using namespace nvcuda;

// Debug flag: enable with -DDEBUG_PRINT during compilation
// #define DEBUG_PRINT 1

// K/V dequantization strategy: 0 = direct (safe, default), 1 = LUT (fast, risky)
#ifndef USE_KV_LUT
#define USE_KV_LUT 0
#endif

// cp.async double-buffering: 0 = direct load (baseline), 1 = async prefetch (Stage-1)
#ifndef USE_CP_ASYNC
#define USE_CP_ASYNC 0
#endif

// WMMA for P·V: 0 = scalar accumulation (Stage-1), 1 = WMMA (Stage-2)
#ifndef USE_WMMA_PV
#define USE_WMMA_PV 0
#endif

// NVTX profiling ranges (optional)
#ifdef ENABLE_NVTX
#include <nvToolsExt.h>
#define NVTX_RANGE(name) nvtxRangePushA(name)
#define NVTX_POP() nvtxRangePop()
#else
#define NVTX_RANGE(name)
#define NVTX_POP()
#endif

// --- Tunables (L4 sm_89, full WMMA) ---
#define HEAD_DIM 64
#define TILE_M   32      // Q rows per block (2 WMMA tiles)
#define TILE_N   32      // KV rows per tile (2 WMMA tiles)
#define NUM_WARPS 4      // REDUCED for WMMA (each warp handles one 16×16 tile)
#define THREADS_PER_BLOCK (NUM_WARPS * 32)
#define D_PAD    64      // No padding needed for WMMA (64 is 16-aligned)

// WMMA tile size
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

// Warp reductions (FP32)
__device__ __forceinline__ float warp_reduce_sum(float v){
    #pragma unroll
    for (int o=16; o>0; o>>=1) v += __shfl_down_sync(0xffffffff, v, o);
    return v;
}

__device__ __forceinline__ float warp_reduce_max(float v){
    #pragma unroll
    for (int o=16; o>0; o>>=1) v = fmaxf(v, __shfl_down_sync(0xffffffff, v, o));
    return v;
}

// Sim-FP8 dequant (symmetric, zero maps exactly to 0)
__device__ __forceinline__ float dequant_sim_fp8(uint8_t u, float s){
    constexpr float INV_MAX = 1.0f / 127.0f;
    float centered = (static_cast<float>(static_cast<int>(u) - 128)) * INV_MAX;
    return centered * 448.0f * s;
}

__launch_bounds__(THREADS_PER_BLOCK, 4)
__global__ void sdpa_fp8_stage_c_wmma_kernel(
    const uint8_t* __restrict__ Q,   // [B,H,S,D]
    const uint8_t* __restrict__ K,   // [B,H,S,D]
    const uint8_t* __restrict__ V,   // [B,H,S,D]
    const float*   __restrict__ Qs,  // [H]
    const float*   __restrict__ Ks,  // [H]
    const float*   __restrict__ Vs,  // [H]
    half*          __restrict__ O,   // [B,H,S,D]
    int B, int H, int S, int D, float softmax_scale
){
    const int b = blockIdx.z;
    const int h = blockIdx.y;
    const int q_block = blockIdx.x;
    const int tid = threadIdx.x;
    const int warp_id = tid >> 5;
    const int lane    = tid & 31;

    const int q_start = q_block * TILE_M;
    const int q_end   = min(q_start + TILE_M, S);
    const int rows_in_tile = q_end - q_start;
    if (rows_in_tile <= 0) return;

    const size_t BHSD = (size_t)S * D;
    const uint8_t* Qbh = Q + ((size_t)b * H + h) * BHSD;
    const uint8_t* Kbh = K + ((size_t)b * H + h) * BHSD;
    const uint8_t* Vbh = V + ((size_t)b * H + h) * BHSD;
    half*         Obh  = O + ((size_t)b * H + h) * BHSD;

    const float q_s = Qs[h], k_s = Ks[h], v_s = Vs[h];

    // --- Shared memory ---
    // Q: row-major [TILE_M][D]
    // K: col-major [D][TILE_N] (transposed for WMMA)
    // SMEM layout for WMMA:
    // - Q: row-major [TILE_M][D_PAD] for matrix_a
    // - K^T: stored as [TILE_N][D_PAD] so elements along D are contiguous (col-major for WMMA matrix_b)
    // - V: row-major [TILE_N][D_PAD]
    __shared__ alignas(16) half sQ[TILE_M][D_PAD];     // 4 KB, row-major
    __shared__ alignas(16) half sKT[TILE_N][D_PAD];    // 4 KB, stored [n][d] for col-major WMMA
    __shared__ alignas(16) half sV[TILE_N][D_PAD];     // 4 KB, row-major
    
#if USE_CP_ASYNC
    // Double-buffering for cp.async prefetch (uint8 staging)
    __shared__ alignas(16) uint8_t sK_u8[2][TILE_N][D_PAD];  // 8 KB (2 buffers)
    __shared__ alignas(16) uint8_t sV_u8[2][TILE_N][D_PAD];  // 8 KB (2 buffers)
#endif
    
#if USE_KV_LUT
    __shared__ float kLUT[256];  // K dequant lookup (1 KB)
    __shared__ float vLUT[256];  // V dequant lookup (1 KB)
#endif
    __shared__ alignas(16) half sS[TILE_M][TILE_N];  // Scores for softmax (2 KB) - MUST be outer scope!
    __shared__ float m_smem[TILE_M];
    __shared__ float l_smem[TILE_M];
    __shared__ alignas(16) float U_smem[TILE_M][D_PAD];  // 8 KB
    
#if USE_WMMA_PV
    #if !defined(USE_FUSED_SOFTMAX_PV) || USE_FUSED_SOFTMAX_PV == 0
    // P tile (unnormalized exp weights for current KV tile): [TILE_M][TILE_N], half
    // Stage-2: Separate sP buffer
    // Stage-3A+: Reuse sS for P (saves 2 KB)
    __shared__ alignas(16) half sP[TILE_M][TILE_N];     // +2 KB
    #endif
    
    // Per-warp scratch to store 16x16 WMMA accumulator (float) before adding into U_smem
    // NUM_WARPS is 4; each 16x16x4B = 1 KB → total +4 KB
    __shared__ alignas(16) float sPV_frag[NUM_WARPS][WMMA_M][WMMA_N]; // +4 KB
#endif

    // Total SMEM: 
    //   USE_WMMA_PV=0: USE_KV_LUT ? 24.5 KB : 22.5 KB (direct dequant saves 2 KB)
    //   USE_WMMA_PV=1 + Stage-2: adds +6 KB (sP+sPV_frag) → ~44.5 KB
    //   USE_WMMA_PV=1 + Stage-3A: adds +4 KB (sPV_frag only, sS reused) → ~42.5 KB (saves 2 KB!)

#if USE_KV_LUT
    // --- Build LUTs (legacy path, requires debugging) ---
    if (tid < 256) {
        const int u = tid;
        constexpr float INV_MAX = 1.0f / 127.0f;
        float centered = (static_cast<float>(u) - 128.0f) * INV_MAX;
        float decoded = centered * 448.0f;
        kLUT[u] = decoded * k_s;
        vLUT[u] = decoded * v_s;
    }
    __syncthreads();  // Ensure all threads see complete LUT before usage

#ifdef DEBUG_PRINT
    if (tid == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0) {
        printf("[DEBUG] USE_KV_LUT=1 (legacy path)\n");
        printf("[DEBUG] LUT addrs: kLUT=%p vLUT=%p sS=%p\n", kLUT, vLUT, sS);
        constexpr float INV_MAX = 1.0f / 127.0f;
        float centered_133 = (133.0f - 128.0f) * INV_MAX;
        float decoded_133 = centered_133 * 448.0f;
        float expected_k133 = decoded_133 * k_s;
        
        float centered_171 = (171.0f - 128.0f) * INV_MAX;
        float decoded_171 = centered_171 * 448.0f;
        float expected_v171 = decoded_171 * v_s;
        
        printf("[DEBUG] Scales: q_s=%.6f k_s=%.6f v_s=%.6f\n", q_s, k_s, v_s);
        printf("[DEBUG] Expected: kLUT[133]=%.4f vLUT[171]=%.4f\n", expected_k133, expected_v171);
        printf("[DEBUG] Actual:   kLUT[133]=%.4f vLUT[171]=%.4f\n", kLUT[133], vLUT[171]);
    }
#endif
#else
    // --- Direct dequant (safe default, no LUT) ---
#ifdef DEBUG_PRINT
    if (tid == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0) {
        printf("[DEBUG] USE_KV_LUT=0 (direct dequant - safe path)\n");
        printf("[DEBUG] Scales: q_s=%.6f k_s=%.6f v_s=%.6f\n", q_s, k_s, v_s);
    }
#endif
#endif

    // --- Load Q tile (uint8→FP16, row-major) ---
    for (int idx = tid; idx < rows_in_tile * D; idx += blockDim.x) {
        int r = idx / D;
        int d = idx % D;
        uint8_t q_u8 = Qbh[(size_t)(q_start + r) * D + d];
        float f = dequant_sim_fp8(q_u8, q_s);
        sQ[r][d] = __float2half(f);
    }

#ifdef DEBUG_PRINT
    __syncthreads();
    if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && tid == 0) {
        printf("[DEBUG] Q tile loaded (row 0, d=0:5): ");
        for (int d = 0; d < 5; d++) {
            printf("%.4f ", __half2float(sQ[0][d]));
        }
        printf("\n");
    }
#endif

    // Init stats and U
    for (int idx = tid; idx < rows_in_tile * D; idx += blockDim.x) {
        int r = idx / D;
        int d = idx % D;
        U_smem[r][d] = 0.f;
    }
    for (int r = tid; r < rows_in_tile; r += blockDim.x) {
        m_smem[r] = -INFINITY;
        l_smem[r] = 0.f;
    }
    __syncthreads();

    const int nTiles = (S + TILE_N - 1) / TILE_N;

#if USE_CP_ASYNC
    // ==========================================
    // cp.async Double-Buffered Pipeline
    // ==========================================
    NVTX_RANGE("KV_loop_cp_async");
    
    // Helper: async copy one tile of K/V (uint8) from gmem to smem staging buffer
    auto cp_async_tile_u8 = [&](int tile_idx, int stage) {
        if (tile_idx >= nTiles) return;
        
        const int kv_start = tile_idx * TILE_N;
        const int kv_len = min(TILE_N, S - kv_start);
        
        constexpr int BYTES = 16;  // 16B chunks for cp.async
        const size_t elems = (size_t)kv_len * D;
        const size_t bytes = elems * sizeof(uint8_t);
        
        uint8_t* __restrict__ dstK = &sK_u8[stage][0][0];
        uint8_t* __restrict__ dstV = &sV_u8[stage][0][0];
        const uint8_t* __restrict__ srcK = Kbh + (size_t)kv_start * D;
        const uint8_t* __restrict__ srcV = Vbh + (size_t)kv_start * D;
        
        // Copy in 16B chunks (safe for cp.async alignment)
        for (size_t off = threadIdx.x * BYTES; off + BYTES <= bytes; off += blockDim.x * BYTES) {
            __pipeline_memcpy_async(dstK + off, srcK + off, BYTES);
            __pipeline_memcpy_async(dstV + off, srcV + off, BYTES);
        }
        
        // Handle tail bytes with scalar copy (fallback for unaligned remainder)
        size_t tail = bytes % BYTES;
        if (tail && threadIdx.x == 0) {
            size_t off_tail = bytes - tail;
            for (size_t i = 0; i < tail; ++i) {
                dstK[off_tail + i] = srcK[off_tail + i];
                dstV[off_tail + i] = srcV[off_tail + i];
            }
        }
        __pipeline_commit();
    };
    
    // Prefetch tile 0 into stage 0
    cp_async_tile_u8(0, 0);
    
    for (int t = 0; t < nTiles; ++t) {
        const int read_stage  = t & 1;
        const int write_stage = (t + 1) & 1;
        
        NVTX_RANGE("tile_iter");
        
        // Prefetch next tile (overlaps with compute below)
        if (t + 1 < nTiles) {
            cp_async_tile_u8(t + 1, write_stage);
        }
        
        // Wait for current tile data (read_stage) to be visible
        __pipeline_wait_prior(1);
        __syncthreads();
        
        // Compute tile bounds
        const int kv_start = t * TILE_N;
        const int kv_len   = min(TILE_N, S - kv_start);
        
        NVTX_RANGE("u8_to_half_dequant");
        // Dequantize from u8 staging buffer → half working buffers (sKT, sV)
        for (int idx = tid; idx < kv_len * D; idx += blockDim.x) {
            int n = idx / D;
            int d = idx % D;
            uint8_t ku = sK_u8[read_stage][n][d];
            uint8_t vu = sV_u8[read_stage][n][d];
#if USE_KV_LUT
            sKT[n][d] = __float2half(kLUT[ku]);
            sV[n][d]  = __float2half(vLUT[vu]);
#else
            float kf = dequant_sim_fp8(ku, k_s);
            float vf = dequant_sim_fp8(vu, v_s);
            sKT[n][d] = __float2half(kf);
            sV[n][d]  = __float2half(vf);
#endif
        }
        
        // Zero-pad for partial tiles
        for (int idx = tid + kv_len * D; idx < TILE_N * D; idx += blockDim.x) {
            int n = idx / D;
            int d = idx % D;
            sKT[n][d] = __float2half(0.f);
            sV[n][d]  = __float2half(0.f);
        }
        NVTX_POP();  // u8_to_half_dequant
        __syncthreads();

#ifdef DEBUG_PRINT
        if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && tid == 0 && t == 0) {
            // Print sQ (Q row 0)
            printf("[DEBUG] sQ[0][0:8]: ");
            for (int d = 0; d < 8; d++) {
                printf("%.4f ", __half2float(sQ[0][d]));
            }
            printf("\n");
            
            // Print sKT (K row 0, stored as sKT[0][d])
            printf("[DEBUG] sKT[0][0:8] (K[0] in col-major view): ");
            for (int d = 0; d < 8; d++) {
                printf("%.4f ", __half2float(sKT[0][d]));
            }
            printf("\n");
            
            // Manual dot product Q[0] @ K[0]
            float manual_dot = 0.0f;
            for (int d = 0; d < D; d++) {
                manual_dot += __half2float(sQ[0][d]) * __half2float(sKT[0][d]);
            }
            printf("[DEBUG] Manual Q[0]@K[0] raw=%.4f (expect ~6.06)\n", manual_dot);
            
            // Print V for reference
            printf("[DEBUG] sV[0][0:8]: ");
            for (int d = 0; d < 8; d++) {
                printf("%.4f ", __half2float(sV[0][d]));
            }
            printf("\n");
        }
#endif

        // =========================================
        // WMMA COMPUTE: Q @ K^T → S (32×32)
        // =========================================
        NVTX_RANGE("WMMA_QK");
        // Each warp handles one 16×16 output tile
        // 4 warps cover 2×2 = 32×32 output
        
        // Warp mapping:
        // warp_id=0 → S[0:16,  0:16]
        // warp_id=1 → S[0:16, 16:32]
        // warp_id=2 → S[16:32, 0:16]
        // warp_id=3 → S[16:32,16:32]
        
        const int warp_m = (warp_id / 2) * WMMA_M;  // 0 or 16
        const int warp_n = (warp_id % 2) * WMMA_N;  // 0 or 16
        
        // Guard partial tiles: skip out-of-range WMMA work
        const bool warp_m_valid = warp_m < rows_in_tile;
        const bool warp_n_valid = warp_n < kv_len;

        // WMMA fragments (FP32 accumulator for better numeric stability)
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;  // FP32 accumulator

        // Zero accumulator
        wmma::fill_fragment(c_frag, 0.0f);

        // Compute Q@K^T only for valid tiles (sS now in outer scope to avoid SMEM aliasing)
        if (warp_m_valid && warp_n_valid) {
            // Compute Q@K^T in 16×16×16 chunks (4 chunks for D=64)
            #pragma unroll
            for (int k = 0; k < D; k += WMMA_K) {
                // Load A: Q[warp_m:warp_m+16, k:k+16] (row-major)
                wmma::load_matrix_sync(a_frag, &sQ[warp_m][k], D_PAD);
                
                // Load B: K^T for col-major WMMA
                // sKT stored as [n][d], so &sKT[col][row] with ldm=D_PAD gives col-major addressing
                // Pointer: &sKT[warp_n][k] = base + warp_n*D_PAD + k
                // WMMA col-major expects: element(row,col) = ptr[row + col*ldm]
                // With ptr=&sKT[warp_n][k], element(r,c) = ptr[r + c*D_PAD] = sKT[warp_n + c][k + r] ✓
                wmma::load_matrix_sync(b_frag, &sKT[warp_n][k], D_PAD);
                
                // MMA: C += A * B
                wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            }

            // Convert FP32 accumulator to FP16 for storage
            wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag_fp16;
            #pragma unroll
            for (int i = 0; i < c_frag.num_elements; i++) {
                c_frag_fp16.x[i] = __float2half(c_frag.x[i]);
            }
            
            // Store result to shared memory
            wmma::store_matrix_sync(&sS[warp_m][warp_n], c_frag_fp16, TILE_N, wmma::mem_row_major);
        }
        NVTX_POP();  // WMMA_QK
        __syncthreads();

#ifdef DEBUG_PRINT
        if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && tid == 0 && t == 0) {
            printf("[DEBUG] Q@K^T raw scores (row 0, n=0:5): ");
            for (int n = 0; n < 5; n++) {
                printf("%.4f ", __half2float(sS[0][n]));
            }
            printf("\n");
            printf("[DEBUG] Q@K^T after scale (row 0, n=0:5, scale=%.6f): ", softmax_scale);
            for (int n = 0; n < 5; n++) {
                printf("%.4f ", __half2float(sS[0][n]) * softmax_scale);
            }
            printf("\n");
        }
#endif

        // =========================================
        // ONLINE SOFTMAX (per row, scalar path)
        // =========================================
        NVTX_RANGE("Softmax_PV");
        // Each warp handles 32/4 = 8 rows
        for (int r = warp_id; r < rows_in_tile; r += NUM_WARPS) {
            // PRIORITY 1 FIX: Each lane loads ALL scores (no stride, no broadcast)
            // Previous bug: Only lane N loaded S_row[N], leaving most elements uninitialized
            // Correct: Each lane loads full S_row[] sequentially → all lanes see same data
            float S_row[TILE_N];
            #pragma unroll
            for (int n = 0; n < kv_len; ++n) {
                S_row[n] = __half2float(sS[r][n]) * softmax_scale;
            }

            // Online softmax update
            float m_old = m_smem[r];
            float m_new = m_old;
            #pragma unroll
            for (int n = 0; n < kv_len; ++n) {
                m_new = fmaxf(m_new, S_row[n]);
            }

            float l_old = l_smem[r];
            float l_add = 0.f;
            #pragma unroll
            for (int n = 0; n < kv_len; ++n) {
                S_row[n] = __expf(S_row[n] - m_new);
                l_add += S_row[n];
            }

            float rescale = __expf(m_old - m_new);
            float l_new = l_old * rescale + l_add;

#ifdef DEBUG_PRINT
            if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && r == 0 && warp_id == 0 && lane == 0 && t == 0) {
                printf("[DEBUG] Softmax (row 0): m_old=%.4f m_new=%.4f l_old=%.4f l_add=%.4f rescale=%.4f\n",
                       m_old, m_new, l_old, l_add, rescale);
                printf("[DEBUG] Attention weights P[0:5]: ");
                for (int n = 0; n < 5; n++) {
                    printf("%.4f ", S_row[n]);
                }
                printf("\n");
            }
#endif

            // Scale U
            for (int d = lane; d < D; d += 32) {
                U_smem[r][d] *= rescale;
            }

#if USE_WMMA_PV
            // Store unnormalized P to shared memory for WMMA P·V
            // Stage-2: sP (separate buffer)
            // Stage-3A+: sS (reuse score buffer, saves 2 KB SMEM)
            for (int n = 0; n < kv_len; ++n) {
                #if defined(USE_FUSED_SOFTMAX_PV) && USE_FUSED_SOFTMAX_PV >= 1
                sS[r][n] = __float2half(S_row[n]);  // Stage-3A: Reuse sS for P
                #else
                sP[r][n] = __float2half(S_row[n]);  // Stage-2: Separate sP
                #endif
            }
            // Zero-pad for partial tiles
            for (int n = kv_len; n < TILE_N; ++n) {
                #if defined(USE_FUSED_SOFTMAX_PV) && USE_FUSED_SOFTMAX_PV >= 1
                sS[r][n] = __float2half(0.f);
                #else
                sP[r][n] = __float2half(0.f);
                #endif
            }
#else
            // Scalar P·V accumulation (Stage-1 path)
            for (int n = 0; n < kv_len; ++n) {
                float p = S_row[n];
                for (int d = lane; d < D; d += 32) {
                    float v = __half2float(sV[n][d]);
                    U_smem[r][d] += p * v;
                }
            }
#endif

            if (lane == 0) {
                m_smem[r] = m_new;
                l_smem[r] = l_new;
            }
        }
        NVTX_POP();  // Softmax_PV
        
#if USE_WMMA_PV
        // Synchronize to ensure sP is visible to all warps
        __syncthreads();
        
        // =========================================
        // WMMA P·V ACCUMULATION (row-major × row-major)
        // =========================================
        NVTX_RANGE("WMMA_PV");
        
        // Warp mapping for U tile (TILE_M x D):
        //   pv_warp_m = (warp_id / 2) * 16   // 0 or 16
        //   dTile starts at (warp_id % 2) and strides by 2 to cover D/16=4 tiles → {0,2} or {1,3}
        const int pv_warp_m = (warp_id / 2) * WMMA_M;
        for (int dTile = (warp_id % 2); dTile < D / WMMA_N; dTile += 2) {
            wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
            wmma::fill_fragment(c_frag, 0.0f);

            // Accumulate over KV dimension (TILE_N) in steps of 16
            #pragma unroll 1  // Reduce register pressure
            for (int kTile = 0; kTile < TILE_N; kTile += WMMA_K) {
                // Guard partial kv_len with zero padding already done for sV and sP

                // A = P[pv_warp_m:pv_warp_m+16, kTile:kTile+16]  (row-major, ldm = TILE_N)
                // Stage-2: Load from sP; Stage-3A+: Load from sS
                wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
                #if defined(USE_FUSED_SOFTMAX_PV) && USE_FUSED_SOFTMAX_PV >= 1
                wmma::load_matrix_sync(a_frag, &sS[pv_warp_m][kTile], TILE_N);  // Stage-3A: Load from sS
                #else
                wmma::load_matrix_sync(a_frag, &sP[pv_warp_m][kTile], TILE_N);  // Stage-2: Load from sP
                #endif

                // B = V[kTile:kTile+16, dTile*16:(dTile+1)*16]  (row-major, ldm = D_PAD)
                wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
                wmma::load_matrix_sync(b_frag, &sV[kTile][dTile * WMMA_N], D_PAD);

                // C += A * B
                wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            }

            // Store C (float) into per-warp scratch and add to U_smem (float), no overlap among warps
            wmma::store_matrix_sync(&sPV_frag[warp_id][0][0], c_frag, WMMA_N, wmma::mem_row_major);
            __syncwarp();

            // Distribute the 16x16 add among lanes
            for (int i = lane; i < WMMA_M * WMMA_N; i += 32) {
                int r_local = i / WMMA_N;     // 0..15
                int d_local = i % WMMA_N;     // 0..15
                int r_glob  = pv_warp_m + r_local;
                int d_glob  = dTile * WMMA_N + d_local;

                if (r_glob < rows_in_tile) {
                    U_smem[r_glob][d_glob] += sPV_frag[warp_id][r_local][d_local];
                }
            }
            __syncwarp();
        }
        NVTX_POP(); // WMMA_PV
#endif
        
        NVTX_POP();  // tile_iter
        __syncthreads();
    }
    
    // Ensure all outstanding cp.async operations are complete
    __pipeline_wait_prior(0);
    __syncthreads();
    NVTX_POP();  // KV_loop_cp_async

#else
    // ==========================================
    // Baseline: Direct Load (no cp.async)
    // ==========================================
    NVTX_RANGE("KV_loop_direct");
    
    for (int t = 0; t < nTiles; ++t) {
        const int kv_start = t * TILE_N;
        const int kv_end   = min(kv_start + TILE_N, S);
        const int kv_len   = kv_end - kv_start;

        NVTX_RANGE("load_KV");
        // --- Load K tile (uint8→FP16, stored as [n][d] for col-major WMMA) ---
        // Store K[n][d] directly to sKT[n][d] - elements along d contiguous = col-major view
        for (int idx = tid; idx < kv_len * D; idx += blockDim.x) {
            int n = idx / D;
            int d = idx % D;
            uint8_t k_u8 = Kbh[(size_t)(kv_start + n) * D + d];
#if USE_KV_LUT
            sKT[n][d] = __float2half(kLUT[k_u8]);  // LUT path
#else
            float k_f = dequant_sim_fp8(k_u8, k_s);  // Direct dequant (safe)
            sKT[n][d] = __float2half(k_f);
#endif
        }

        // --- Load V tile (uint8→FP16, row-major) ---
        for (int idx = tid; idx < kv_len * D; idx += blockDim.x) {
            int n = idx / D;
            int d = idx % D;
            uint8_t v_u8 = Vbh[(size_t)(kv_start + n) * D + d];
#if USE_KV_LUT
            sV[n][d] = __float2half(vLUT[v_u8]);  // LUT path
#else
            float v_f = dequant_sim_fp8(v_u8, v_s);  // Direct dequant (safe)
            sV[n][d] = __float2half(v_f);
#endif
        }
        
        // Zero-pad K^T/V for partial tiles (kv_len < TILE_N)
        for (int idx = tid + kv_len * D; idx < TILE_N * D; idx += blockDim.x) {
            int n = idx / D;
            int d = idx % D;
            sKT[n][d] = __float2half(0.f);  // Match new [n][d] layout
            sV[n][d]  = __float2half(0.f);
        }
        NVTX_POP();  // load_KV
        __syncthreads();

#ifdef DEBUG_PRINT
        if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && tid == 0 && t == 0) {
            // Print sQ (Q row 0)
            printf("[DEBUG] sQ[0][0:8]: ");
            for (int d = 0; d < 8; d++) {
                printf("%.4f ", __half2float(sQ[0][d]));
            }
            printf("\n");
            
            // Print sKT (K row 0, stored as sKT[0][d])
            printf("[DEBUG] sKT[0][0:8] (K[0] in col-major view): ");
            for (int d = 0; d < 8; d++) {
                printf("%.4f ", __half2float(sKT[0][d]));
            }
            printf("\n");
            
            // Manual dot product Q[0] @ K[0]
            float manual_dot = 0.0f;
            for (int d = 0; d < D; d++) {
                manual_dot += __half2float(sQ[0][d]) * __half2float(sKT[0][d]);
            }
            printf("[DEBUG] Manual Q[0]@K[0] raw=%.4f (expect ~6.06)\n", manual_dot);
            
            // Print V for reference
            printf("[DEBUG] sV[0][0:8]: ");
            for (int d = 0; d < 8; d++) {
                printf("%.4f ", __half2float(sV[0][d]));
            }
            printf("\n");
        }
#endif

        // =========================================
        // WMMA COMPUTE: Q @ K^T → S (32×32)
        // =========================================
        NVTX_RANGE("WMMA_QK");
        const int warp_m = (warp_id / 2) * WMMA_M;
        const int warp_n = (warp_id % 2) * WMMA_N;
        const bool warp_m_valid = warp_m < rows_in_tile;
        const bool warp_n_valid = warp_n < kv_len;

        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
        wmma::fill_fragment(c_frag, 0.0f);

        if (warp_m_valid && warp_n_valid) {
            #pragma unroll
            for (int k = 0; k < D; k += WMMA_K) {
                wmma::load_matrix_sync(a_frag, &sQ[warp_m][k], D_PAD);
                wmma::load_matrix_sync(b_frag, &sKT[warp_n][k], D_PAD);
                wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            }
            wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag_fp16;
            #pragma unroll
            for (int i = 0; i < c_frag.num_elements; i++) {
                c_frag_fp16.x[i] = __float2half(c_frag.x[i]);
            }
            wmma::store_matrix_sync(&sS[warp_m][warp_n], c_frag_fp16, TILE_N, wmma::mem_row_major);
        }
        NVTX_POP();  // WMMA_QK
        __syncthreads();

#ifdef DEBUG_PRINT
        if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && tid == 0 && t == 0) {
            printf("[DEBUG] Q@K^T raw scores (row 0, n=0:5): ");
            for (int n = 0; n < 5; n++) {
                printf("%.4f ", __half2float(sS[0][n]));
            }
            printf("\n");
            printf("[DEBUG] Q@K^T after scale (row 0, n=0:5, scale=%.6f): ", softmax_scale);
            for (int n = 0; n < 5; n++) {
                printf("%.4f ", __half2float(sS[0][n]) * softmax_scale);
            }
            printf("\n");
        }
#endif

        // =========================================
        // ONLINE SOFTMAX (per row, scalar path)
        // =========================================
        NVTX_RANGE("Softmax_PV");
        for (int r = warp_id; r < rows_in_tile; r += NUM_WARPS) {
            float S_row[TILE_N];
            #pragma unroll
            for (int n = 0; n < kv_len; ++n) {
                S_row[n] = __half2float(sS[r][n]) * softmax_scale;
            }

            float m_old = m_smem[r];
            float m_new = m_old;
            #pragma unroll
            for (int n = 0; n < kv_len; ++n) {
                m_new = fmaxf(m_new, S_row[n]);
            }

            float l_old = l_smem[r];
            float l_add = 0.f;
            #pragma unroll
            for (int n = 0; n < kv_len; ++n) {
                S_row[n] = __expf(S_row[n] - m_new);
                l_add += S_row[n];
            }

            float rescale = __expf(m_old - m_new);
            float l_new = l_old * rescale + l_add;

#ifdef DEBUG_PRINT
            if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && r == 0 && warp_id == 0 && lane == 0 && t == 0) {
                printf("[DEBUG] Softmax (row 0): m_old=%.4f m_new=%.4f l_old=%.4f l_add=%.4f rescale=%.4f\n",
                       m_old, m_new, l_old, l_add, rescale);
                printf("[DEBUG] Attention weights P[0:5]: ");
                for (int n = 0; n < 5; n++) {
                    printf("%.4f ", S_row[n]);
                }
                printf("\n");
            }
#endif

            for (int d = lane; d < D; d += 32) {
                U_smem[r][d] *= rescale;
            }

#if USE_WMMA_PV
            // Store unnormalized P to shared memory for WMMA P·V
            // Stage-2: sP (separate buffer)
            // Stage-3A+: sS (reuse score buffer, saves 2 KB SMEM)
            for (int n = 0; n < kv_len; ++n) {
                #if defined(USE_FUSED_SOFTMAX_PV) && USE_FUSED_SOFTMAX_PV >= 1
                sS[r][n] = __float2half(S_row[n]);  // Stage-3A: Reuse sS for P
                #else
                sP[r][n] = __float2half(S_row[n]);  // Stage-2: Separate sP
                #endif
            }
            // Zero-pad for partial tiles
            for (int n = kv_len; n < TILE_N; ++n) {
                #if defined(USE_FUSED_SOFTMAX_PV) && USE_FUSED_SOFTMAX_PV >= 1
                sS[r][n] = __float2half(0.f);
                #else
                sP[r][n] = __float2half(0.f);
                #endif
            }
#else
            // Scalar P·V accumulation (Stage-1 path)
            for (int n = 0; n < kv_len; ++n) {
                float p = S_row[n];
                for (int d = lane; d < D; d += 32) {
                    float v = __half2float(sV[n][d]);
                    U_smem[r][d] += p * v;
                }
            }
#endif

            if (lane == 0) {
                m_smem[r] = m_new;
                l_smem[r] = l_new;
            }
        }
        NVTX_POP();  // Softmax_PV
        
#if USE_WMMA_PV
        // Synchronize to ensure sP is visible to all warps
        __syncthreads();
        
        // =========================================
        // WMMA P·V ACCUMULATION (row-major × row-major)
        // =========================================
        NVTX_RANGE("WMMA_PV");
        
        // Warp mapping for U tile (TILE_M x D):
        //   pv_warp_m = (warp_id / 2) * 16   // 0 or 16
        //   dTile starts at (warp_id % 2) and strides by 2 to cover D/16=4 tiles → {0,2} or {1,3}
        const int pv_warp_m = (warp_id / 2) * WMMA_M;
        for (int dTile = (warp_id % 2); dTile < D / WMMA_N; dTile += 2) {
            wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
            wmma::fill_fragment(c_frag, 0.0f);

            // Accumulate over KV dimension (TILE_N) in steps of 16
            #pragma unroll 1  // Reduce register pressure
            for (int kTile = 0; kTile < TILE_N; kTile += WMMA_K) {
                // Guard partial kv_len with zero padding already done for sV and sP

                // A = P[pv_warp_m:pv_warp_m+16, kTile:kTile+16]  (row-major, ldm = TILE_N)
                // Stage-2: Load from sP; Stage-3A+: Load from sS
                wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
                #if defined(USE_FUSED_SOFTMAX_PV) && USE_FUSED_SOFTMAX_PV >= 1
                wmma::load_matrix_sync(a_frag, &sS[pv_warp_m][kTile], TILE_N);  // Stage-3A: Load from sS
                #else
                wmma::load_matrix_sync(a_frag, &sP[pv_warp_m][kTile], TILE_N);  // Stage-2: Load from sP
                #endif

                // B = V[kTile:kTile+16, dTile*16:(dTile+1)*16]  (row-major, ldm = D_PAD)
                wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
                wmma::load_matrix_sync(b_frag, &sV[kTile][dTile * WMMA_N], D_PAD);

                // C += A * B
                wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            }

            // Store C (float) into per-warp scratch and add to U_smem (float), no overlap among warps
            wmma::store_matrix_sync(&sPV_frag[warp_id][0][0], c_frag, WMMA_N, wmma::mem_row_major);
            __syncwarp();

            // Distribute the 16x16 add among lanes
            for (int i = lane; i < WMMA_M * WMMA_N; i += 32) {
                int r_local = i / WMMA_N;     // 0..15
                int d_local = i % WMMA_N;     // 0..15
                int r_glob  = pv_warp_m + r_local;
                int d_glob  = dTile * WMMA_N + d_local;

                if (r_glob < rows_in_tile) {
                    U_smem[r_glob][d_glob] += sPV_frag[warp_id][r_local][d_local];
                }
            }
            __syncwarp();
        }
        NVTX_POP(); // WMMA_PV
#endif
        
        __syncthreads();
    }
    NVTX_POP();  // KV_loop_direct
#endif

    // --- Write O = U / l ---
    for (int r = warp_id; r < rows_in_tile; r += NUM_WARPS) {
        float l_final = l_smem[r];
        half* out = Obh + (size_t)(q_start + r) * D;
        
#ifdef DEBUG_PRINT
        if (blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0 && r == 0 && warp_id == 0 && lane == 0) {
            printf("[DEBUG] Final output (row 0): l_final=%.4f\n", l_final);
            printf("[DEBUG] U_smem[0][0:5] / l_final: ");
            for (int d = 0; d < 5; d++) {
                printf("%.4f ", U_smem[0][d] / l_final);
            }
            printf("\n");
        }
#endif
        
        for (int d = lane; d < D; d += 32) {
            float o = U_smem[r][d] / l_final;
            out[d] = __float2half(o);
        }
    }
}

extern "C" void launch_sdpa_fp8_stage_c_wmma(
    const void* Q,
    const void* K,
    const void* V,
    const float* Q_scale,
    const float* K_scale,
    const float* V_scale,
    half* O,
    int B, int H, int S, int D,
    float softmax_scale,
    hipStream_t stream
) {
    dim3 grid((S + TILE_M - 1) / TILE_M, H, B);
    dim3 block(THREADS_PER_BLOCK);

    sdpa_fp8_stage_c_wmma_kernel<<<grid, block, 0, stream>>>(
        reinterpret_cast<const uint8_t*>(Q),
        reinterpret_cast<const uint8_t*>(K),
        reinterpret_cast<const uint8_t*>(V),
        Q_scale,
        K_scale,
        V_scale,
        O,
        B, H, S, D,
        softmax_scale
    );
}

