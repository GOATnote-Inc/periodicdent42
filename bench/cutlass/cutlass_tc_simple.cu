#include "hip/hip_runtime.h"
// Minimal CUTLASS TensorOp using defaults
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"

// Let CUTLASS pick sensible defaults for Sm80
using ElementA = cutlass::half_t;
using ElementB = cutlass::half_t;
using ElementC = float;
using ElementAccumulator = float;

using LayoutA = cutlass::layout::RowMajor;
using LayoutB = cutlass::layout::ColumnMajor;
using LayoutC = cutlass::layout::RowMajor;

// Minimal template args - let CUTLASS fill in the rest
using Gemm = cutlass::gemm::device::Gemm<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC, LayoutC,
    ElementAccumulator,
    cutlass::arch::OpClassTensorOp,
    cutlass::arch::Sm80
>;

int main() {
    const int M = 32, N = 32, K = 64;
    
    printf("CUTLASS TensorOp (defaults)\n");
    printf("M=%d, N=%d, K=%d\n", M, N, K);
    
    // Allocate
    ElementA *h_A = (ElementA*)malloc(M * K * sizeof(ElementA));
    ElementB *h_B = (ElementB*)malloc(K * N * sizeof(ElementB));
    ElementC *h_C = (ElementC*)malloc(M * N * sizeof(ElementC));
    
    for (int i = 0; i < M * K; i++) h_A[i] = ElementA(0.1f);
    for (int i = 0; i < K * N; i++) h_B[i] = ElementB(0.1f);
    for (int i = 0; i < M * N; i++) h_C[i] = 0.0f;
    
    ElementA *d_A;
    ElementB *d_B;
    ElementC *d_C;
    hipMalloc(&d_A, M * K * sizeof(ElementA));
    hipMalloc(&d_B, K * N * sizeof(ElementB));
    hipMalloc(&d_C, M * N * sizeof(ElementC));
    
    hipMemcpy(d_A, h_A, M * K * sizeof(ElementA), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(ElementB), hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, M * N * sizeof(ElementC), hipMemcpyHostToDevice);
    
    // Setup
    typename Gemm::Arguments args(
        {M, N, K},
        {d_A, K},
        {d_B, K},
        {d_C, N},
        {d_C, N},
        {1.0f / sqrtf(K), 0.0f}
    );
    
    Gemm gemm_op;
    
    cutlass::Status status = gemm_op.can_implement(args);
    if (status != cutlass::Status::kSuccess) {
        fprintf(stderr, "❌ can_implement: %s\n", cutlassGetStatusString(status));
        return 1;
    }
    printf("✅ can_implement\n");
    
    size_t workspace_size = gemm_op.get_workspace_size(args);
    void* workspace = nullptr;
    if (workspace_size > 0) {
        hipMalloc(&workspace, workspace_size);
    }
    printf("Workspace: %zu bytes\n", workspace_size);
    
    status = gemm_op.initialize(args, workspace);
    if (status != cutlass::Status::kSuccess) {
        fprintf(stderr, "❌ initialize: %s\n", cutlassGetStatusString(status));
        if (workspace) hipFree(workspace);
        return 1;
    }
    printf("✅ initialize\n");
    
    printf("Launching...\n");
    status = gemm_op();
    
    hipError_t cuda_err = hipGetLastError();
    hipDeviceSynchronize();
    hipError_t sync_err = hipGetLastError();
    
    if (status != cutlass::Status::kSuccess) {
        fprintf(stderr, "❌ launch: %s\n", cutlassGetStatusString(status));
        fprintf(stderr, "   CUDA: %s / %s\n", 
                hipGetErrorString(cuda_err), hipGetErrorString(sync_err));
        if (workspace) hipFree(workspace);
        return 1;
    }
    
    if (sync_err != hipSuccess) {
        fprintf(stderr, "❌ sync: %s\n", hipGetErrorString(sync_err));
        if (workspace) hipFree(workspace);
        return 1;
    }
    
    printf("✅ launch\n");
    
    // Verify
    hipMemcpy(h_C, d_C, M * N * sizeof(ElementC), hipMemcpyDeviceToHost);
    
    float expected = 0.1f * 0.1f * K / sqrtf(K);
    printf("C[0,0]=%.6f (expect ~%.6f)\n", h_C[0], expected);
    
    bool ok = (h_C[0] > 0.0f && h_C[0] < 2.0f * expected);
    printf("%s\n", ok ? "✅ PASS" : "❌ FAIL");
    
    // Benchmark if OK
    if (ok) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        hipEventRecord(start);
        for (int i = 0; i < 100; i++) {
            gemm_op();
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        printf("Perf: %.2f μs/iter\n", ms * 10.0f);
        
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    
    // Cleanup
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    if (workspace) hipFree(workspace);
    
    return ok ? 0 : 1;
}

